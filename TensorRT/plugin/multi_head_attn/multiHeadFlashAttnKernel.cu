#include "hip/hip_runtime.h"
//
// Created by Derry Lin on 2023/6/21.
//

#include "cuda_helper.h"
#include "multiHeadFlashAttnKernel.h"
#include <cmath>
#include <cuda/std/limits>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <unistd.h>

#define TC_SIZE 16
#define WARP_SIZE 32

template <typename T>
__forceinline__ __device__ T hmax(const T &a, const T &b) {
  return max(a, b);
}

#if __CUDA_ARCH__ >= 800
template <>
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) {
  return __hmax(a, b);
}
template <>
__forceinline__ __device__ __half2 hmax(const __half2 &a, const __half2 &b) {
  return __hmax2(a, b);
}
#else
template <>
__forceinline__ __device__ __half hmax(const __half &a, const __half &b) {
  return __hgt(a, b) ? a : b;
}
template <>
__forceinline__ __device__ __half2 hmax(const __half2 &a, const __half2 &b) {
  return __hfma2(__hgt2(a, b), a, __hmul2(__hle2(a, b), b));
}
#endif

template <int HEAD_DIM, int BASE_SEQ_LEN>
__global__ void FMHAInferKernel(const float *__restrict__ query,
                                const float *__restrict__ key,
                                const float *__restrict__ value, float sqrt_d,
                                float *output, const int KV_LEN) {
  static_assert(BASE_SEQ_LEN % TC_SIZE == 0 && HEAD_DIM % TC_SIZE == 0 &&
                    BASE_SEQ_LEN >= HEAD_DIM,
                "");
  const int NUM_WARPS = BASE_SEQ_LEN / TC_SIZE;
  const int NUM_HEAD_WARPS = HEAD_DIM / TC_SIZE;

  __shared__ __half share_buffer[BASE_SEQ_LEN * TC_SIZE * 5];
  auto qk_buffer_f =
      reinterpret_cast<float *>(share_buffer + BASE_SEQ_LEN * TC_SIZE * 2);
  auto qk_buffer_h =
      reinterpret_cast<__half2 *>(share_buffer + BASE_SEQ_LEN * TC_SIZE * 4);

  const unsigned int batch = blockIdx.y;
  const unsigned int Q_LEN = gridDim.x * BASE_SEQ_LEN;
  const unsigned int q_start = blockIdx.x * BASE_SEQ_LEN;
  const unsigned int tid = threadIdx.x;
  const unsigned int warp_id = tid / WARP_SIZE;
  const unsigned int lane_id = tid % WARP_SIZE;
  const unsigned int warp_num = blockDim.x / WARP_SIZE;

  const unsigned int mem_lane_id_x = lane_id % 8;
  const unsigned int mem_lane_id_y = lane_id / 8;

  __half *smem_q[2];
  __half *smem_k[2];
  __half *smem_v[2];

  smem_q[0] = share_buffer;
  smem_q[1] = share_buffer + 1 * BASE_SEQ_LEN * TC_SIZE;
  smem_k[0] = share_buffer + 2 * BASE_SEQ_LEN * TC_SIZE;
  smem_k[1] = share_buffer + 3 * BASE_SEQ_LEN * TC_SIZE;
  smem_v[0] = share_buffer + 2 * BASE_SEQ_LEN * TC_SIZE;
  smem_v[1] = share_buffer + 3 * BASE_SEQ_LEN * TC_SIZE;

  const float *query_ptr =
      query + batch * Q_LEN * HEAD_DIM + q_start * HEAD_DIM;
  float *output_ptr = output + batch * Q_LEN * HEAD_DIM + q_start * HEAD_DIM;

  float thread_max_old[2] = {-cuda::std::numeric_limits<float>::infinity(),
                             -cuda::std::numeric_limits<float>::infinity()};
  float thread_sum_old[2] = {0, 0};

  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half, nvcuda::wmma::row_major>
      q_frag;
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half, nvcuda::wmma::col_major>
      k_frag;
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half, nvcuda::wmma::row_major>
      qk_frag[NUM_WARPS];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half, nvcuda::wmma::row_major>
      v_frag;
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, TC_SIZE, TC_SIZE, TC_SIZE,
                         float>
      kv_out_frag[NUM_WARPS > NUM_HEAD_WARPS ? NUM_WARPS : NUM_HEAD_WARPS];
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, TC_SIZE, TC_SIZE, TC_SIZE,
                         float>
      out_frag[NUM_HEAD_WARPS];

#pragma unroll
  for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
    nvcuda::wmma::fill_fragment(out_frag[xi], 0.f);
  }

#pragma unroll
  for (int kv_start = 0; kv_start < KV_LEN; kv_start += BASE_SEQ_LEN) {
    const float *key_ptr =
        key + batch * KV_LEN * HEAD_DIM + kv_start * HEAD_DIM;

    float thread_max[2] = {-cuda::std::numeric_limits<float>::infinity(),
                           -cuda::std::numeric_limits<float>::infinity()};
    float thread_sum[2] = {0, 0};

#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
      nvcuda::wmma::fill_fragment(kv_out_frag[xi], 0.f);
    }

    int k = 0, stride_warp = TC_SIZE * TC_SIZE / 2 / WARP_SIZE;
    float2 f2;
    __half2 h2;
    for (int i = 0; i < stride_warp; i++) {
      f2 = *reinterpret_cast<const float2 *>(
          query_ptr + warp_id * TC_SIZE * HEAD_DIM +
          i * 2 * WARP_SIZE * HEAD_DIM / TC_SIZE + mem_lane_id_y * HEAD_DIM +
          mem_lane_id_x * 2 + k * TC_SIZE);
      h2 = __float22half2_rn(f2);
      *(__half2 *)(smem_q[k % 2] + warp_id * TC_SIZE * TC_SIZE +
                   i * WARP_SIZE * 2 + mem_lane_id_y * TC_SIZE +
                   mem_lane_id_x * 2) = h2;

      f2 = *reinterpret_cast<const float2 *>(
          key_ptr + warp_id * TC_SIZE * HEAD_DIM +
          i * 2 * WARP_SIZE * HEAD_DIM / TC_SIZE + mem_lane_id_y * HEAD_DIM +
          mem_lane_id_x * 2 + k * TC_SIZE);
      h2 = __float22half2_rn(f2);
      *(__half2 *)(smem_k[k % 2] + warp_id * TC_SIZE * TC_SIZE +
                   i * WARP_SIZE * 2 + mem_lane_id_y * TC_SIZE +
                   mem_lane_id_x * 2) = h2;
    }

    for (k = 1; k < NUM_HEAD_WARPS; k++) {
      __syncthreads();
      for (int i = 0; i < stride_warp; i++) {
        f2 = *reinterpret_cast<const float2 *>(
            query_ptr + warp_id * TC_SIZE * HEAD_DIM +
            i * 2 * WARP_SIZE * HEAD_DIM / TC_SIZE + mem_lane_id_y * HEAD_DIM +
            mem_lane_id_x * 2 + k * TC_SIZE);
        h2 = __float22half2_rn(f2);
        *(__half2 *)(smem_q[k % 2] + warp_id * TC_SIZE * TC_SIZE +
                     i * WARP_SIZE * 2 + mem_lane_id_y * TC_SIZE +
                     mem_lane_id_x * 2) = h2;

        f2 = *reinterpret_cast<const float2 *>(
            key_ptr + warp_id * TC_SIZE * HEAD_DIM +
            i * 2 * WARP_SIZE * HEAD_DIM / TC_SIZE + mem_lane_id_y * HEAD_DIM +
            mem_lane_id_x * 2 + k * TC_SIZE);
        h2 = __float22half2_rn(f2);
        *(__half2 *)(smem_k[k % 2] + warp_id * TC_SIZE * TC_SIZE +
                     i * WARP_SIZE * 2 + mem_lane_id_y * TC_SIZE +
                     mem_lane_id_x * 2) = h2;
      }

      nvcuda::wmma::load_matrix_sync(
          q_frag, &(smem_q[(k - 1) % 2][warp_id * TC_SIZE * TC_SIZE]), TC_SIZE);
      for (int xi = 0; xi < NUM_WARPS; xi++) {
        nvcuda::wmma::load_matrix_sync(
            k_frag, &(smem_k[(k - 1) % 2][xi * TC_SIZE * TC_SIZE]), TC_SIZE);
        nvcuda::wmma::mma_sync(kv_out_frag[xi], q_frag, k_frag,
                               kv_out_frag[xi]);
      }
    }
    __syncthreads();
    k = NUM_HEAD_WARPS - 1;
    nvcuda::wmma::load_matrix_sync(
        q_frag, &(smem_q[k % 2][warp_id * TC_SIZE * TC_SIZE]), TC_SIZE);
    for (int xi = 0; xi < NUM_WARPS; xi++) {
      nvcuda::wmma::load_matrix_sync(
          k_frag, &(smem_k[k % 2][xi * TC_SIZE * TC_SIZE]), TC_SIZE);
      nvcuda::wmma::mma_sync(kv_out_frag[xi], q_frag, k_frag, kv_out_frag[xi]);
    }

#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
#pragma unroll
      for (int tc_yi = 0; tc_yi < 2; tc_yi++) {
#pragma unroll
        for (int tc_xi = 0; tc_xi < 2; tc_xi++) {
          kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0] *= sqrt_d;
          kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1] *= sqrt_d;
          thread_max[tc_yi] =
              max(thread_max[tc_yi],
                  max(kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0],
                      kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1]));
        }
      }
    }

#pragma unroll
    for (int s = 2; s > 0; s >>= 1) {
#pragma unroll
      for (int tc_yi = 0; tc_yi < 2; tc_yi++) {
        thread_max[tc_yi] =
            max(thread_max[tc_yi],
                __shfl_xor_sync(0xffffffff, thread_max[tc_yi], s, 4));
      }
    }

#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
#pragma unroll
      for (int tc_yi = 0; tc_yi < 2; tc_yi++) {
#pragma unroll
        for (int tc_xi = 0; tc_xi < 2; tc_xi++) {
          kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0] = __expf(
              kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0] - thread_max[tc_yi]);
          kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1] = __expf(
              kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1] - thread_max[tc_yi]);
          thread_sum[tc_yi] += (kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0] +
                                kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1]);
        }
      }
    }

#pragma unroll
    for (int s = 2; s > 0; s >>= 1) {
#pragma unroll
      for (int tc_yi = 0; tc_yi < 2; tc_yi++) {
        thread_sum[tc_yi] +=
            __shfl_xor_sync(0xffffffff, thread_sum[tc_yi], s, 4);
      }
    }

    __syncthreads();
#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
      nvcuda::wmma::store_matrix_sync(qk_buffer_f + warp_id * TC_SIZE * TC_SIZE,
                                      kv_out_frag[xi], TC_SIZE,
                                      nvcuda::wmma::mem_row_major);
      for (int i = 0; i < TC_SIZE * TC_SIZE / WARP_SIZE / 2; i++) {
        float t1 = qk_buffer_f[warp_id * TC_SIZE * TC_SIZE + i * WARP_SIZE * 2 +
                               lane_id * 2];
        float t2 = qk_buffer_f[warp_id * TC_SIZE * TC_SIZE + i * WARP_SIZE * 2 +
                               lane_id * 2 + 1];
        __half2 t3 = __floats2half2_rn(t1, t2);
        qk_buffer_h[warp_id * TC_SIZE * TC_SIZE / 2 + i * WARP_SIZE + lane_id] =
            t3;
      }
      nvcuda::wmma::load_matrix_sync(
          qk_frag[xi],
          reinterpret_cast<__half *>(qk_buffer_h +
                                     warp_id * TC_SIZE * TC_SIZE / 2),
          TC_SIZE);
    }
    __syncthreads();

#pragma unroll
    for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
      nvcuda::wmma::fill_fragment(kv_out_frag[xi], 0.f);
    }

    stride_warp = TC_SIZE / warp_num;
    const float *value_ptr = value + batch * KV_LEN * HEAD_DIM +
                             (kv_start + stride_warp * warp_id) * HEAD_DIM;
    k = 0;
    for (int i = 0; i < stride_warp * HEAD_DIM / 2 / WARP_SIZE; i++) {
      f2 = *reinterpret_cast<const float2 *>(
          value_ptr + k * TC_SIZE * HEAD_DIM + i * WARP_SIZE * 2 + lane_id * 2);
      h2 = __float22half2_rn(f2);
      *(__half2 *)(smem_v[k % 2] + warp_id * stride_warp * HEAD_DIM +
                   i * WARP_SIZE * 2 + lane_id * 2) = h2;
    }

    for (k = 1; k < NUM_WARPS; k++) {
      __syncthreads();
      for (int i = 0; i < stride_warp * HEAD_DIM / 2 / WARP_SIZE; i++) {
        f2 = *reinterpret_cast<const float2 *>(value_ptr +
                                               k * TC_SIZE * HEAD_DIM +
                                               i * WARP_SIZE * 2 + lane_id * 2);
        h2 = __float22half2_rn(f2);
        *(__half2 *)(smem_v[k % 2] + warp_id * stride_warp * HEAD_DIM +
                     i * WARP_SIZE * 2 + lane_id * 2) = h2;
      }

      for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
        nvcuda::wmma::load_matrix_sync(
            v_frag, &(smem_v[(k - 1) % 2][xi * TC_SIZE]), HEAD_DIM);
        nvcuda::wmma::mma_sync(kv_out_frag[xi], qk_frag[k - 1], v_frag,
                               kv_out_frag[xi]);
      }
    }
    __syncthreads();
    k = NUM_WARPS - 1;
    for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
      nvcuda::wmma::load_matrix_sync(v_frag, &(smem_v[k % 2][xi * TC_SIZE]),
                                     HEAD_DIM);
      nvcuda::wmma::mma_sync(kv_out_frag[xi], qk_frag[k], v_frag,
                             kv_out_frag[xi]);
    }

#pragma unroll
    for (int tc_yi = 0; tc_yi < 2; tc_yi++) {
      float thread_max_new = max(thread_max_old[tc_yi], thread_max[tc_yi]);
      float exp_max_old = __expf(thread_max_old[tc_yi] - thread_max_new);
      float exp_max = __expf(thread_max[tc_yi] - thread_max_new);
      float thread_sum_new =
          exp_max_old * thread_sum_old[tc_yi] + exp_max * thread_sum[tc_yi];
#pragma unroll
      for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
#pragma unroll
        for (int tc_xi = 0; tc_xi < 2; tc_xi++) {
          out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0] =
              __frcp_rn(thread_sum_new) *
              (thread_sum_old[tc_yi] * exp_max_old *
                   out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0] +
               exp_max * (kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0]));
          out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1] =
              __frcp_rn(thread_sum_new) *
              (thread_sum_old[tc_yi] * exp_max_old *
                   out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1] +
               exp_max * (kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1]));
        }
      }
      thread_sum_old[tc_yi] = thread_sum_new;
      thread_max_old[tc_yi] = thread_max_new;
    }
  }

  for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
    nvcuda::wmma::store_matrix_sync(
        output_ptr + warp_id * TC_SIZE * HEAD_DIM + xi * TC_SIZE, out_frag[xi],
        HEAD_DIM, nvcuda::wmma::mem_row_major);
  }
}

template <int HEAD_DIM, int BASE_SEQ_LEN>
__global__ void FMHAInferKernel(const __half *__restrict__ query,
                                const __half *__restrict__ key,
                                const __half *__restrict__ value, __half sqrt_d,
                                __half *output, const int KV_LEN) {
  static_assert(BASE_SEQ_LEN % TC_SIZE == 0 && HEAD_DIM % TC_SIZE == 0 &&
                    BASE_SEQ_LEN >= HEAD_DIM,
                "");
  const int NUM_WARPS = BASE_SEQ_LEN / TC_SIZE;
  const int NUM_HEAD_WARPS = HEAD_DIM / TC_SIZE;

  __shared__ __half share_buffer[BASE_SEQ_LEN * TC_SIZE * 4];
  auto qk_buffer_fh = share_buffer + BASE_SEQ_LEN * TC_SIZE * 2;

  const unsigned int batch = blockIdx.y;
  const unsigned int Q_LEN = gridDim.x * BASE_SEQ_LEN;
  const unsigned int q_start = blockIdx.x * BASE_SEQ_LEN;
  const unsigned int tid = threadIdx.x;
  const unsigned int warp_id = tid / WARP_SIZE;
  const unsigned int lane_id = tid % WARP_SIZE;
  const unsigned int warp_num = blockDim.x / WARP_SIZE;

  const unsigned int mem_lane_id_x = lane_id % 8;
  const unsigned int mem_lane_id_y = lane_id / 8;

  __half *smem_q[2];
  __half *smem_k[2];
  __half *smem_v[2];

  smem_q[0] = share_buffer;
  smem_q[1] = share_buffer + 1 * BASE_SEQ_LEN * TC_SIZE;
  smem_k[0] = share_buffer + 2 * BASE_SEQ_LEN * TC_SIZE;
  smem_k[1] = share_buffer + 3 * BASE_SEQ_LEN * TC_SIZE;
  smem_v[0] = share_buffer + 2 * BASE_SEQ_LEN * TC_SIZE;
  smem_v[1] = share_buffer + 3 * BASE_SEQ_LEN * TC_SIZE;

  const __half *query_ptr =
      query + batch * Q_LEN * HEAD_DIM + q_start * HEAD_DIM;
  __half *output_ptr = output + batch * Q_LEN * HEAD_DIM + q_start * HEAD_DIM;

  __half thread_max_old[2] = {-cuda::std::numeric_limits<__half>::infinity(),
                              -cuda::std::numeric_limits<__half>::infinity()};
  __half thread_sum_old[2] = {0, 0};

  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half, nvcuda::wmma::row_major>
      q_frag;
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half, nvcuda::wmma::col_major>
      k_frag;
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half, nvcuda::wmma::row_major>
      qk_frag[NUM_WARPS];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half, nvcuda::wmma::row_major>
      v_frag;
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half>
      kv_out_frag[NUM_WARPS > NUM_HEAD_WARPS ? NUM_WARPS : NUM_HEAD_WARPS];
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half>
      out_frag[NUM_HEAD_WARPS];

#pragma unroll
  for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
    nvcuda::wmma::fill_fragment(out_frag[xi], 0.f);
  }

#pragma unroll
  for (int kv_start = 0; kv_start < KV_LEN; kv_start += BASE_SEQ_LEN) {
    const __half *key_ptr =
        key + batch * KV_LEN * HEAD_DIM + kv_start * HEAD_DIM;

    __half thread_max[2] = {-cuda::std::numeric_limits<__half>::infinity(),
                            -cuda::std::numeric_limits<__half>::infinity()};
    __half thread_sum[2] = {0, 0};

#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
      nvcuda::wmma::fill_fragment(kv_out_frag[xi], 0.f);
    }

    int k = 0, stride_warp = TC_SIZE * TC_SIZE / 2 / WARP_SIZE;
    __half2 h2;
    for (int i = 0; i < stride_warp; i++) {
      h2 = *reinterpret_cast<const __half2 *>(
          query_ptr + warp_id * TC_SIZE * HEAD_DIM +
          i * 2 * WARP_SIZE * HEAD_DIM / TC_SIZE + mem_lane_id_y * HEAD_DIM +
          mem_lane_id_x * 2 + k * TC_SIZE);
      *(__half2 *)(smem_q[k % 2] + warp_id * TC_SIZE * TC_SIZE +
                   i * WARP_SIZE * 2 + mem_lane_id_y * TC_SIZE +
                   mem_lane_id_x * 2) = h2;

      h2 = *reinterpret_cast<const __half2 *>(
          key_ptr + warp_id * TC_SIZE * HEAD_DIM +
          i * 2 * WARP_SIZE * HEAD_DIM / TC_SIZE + mem_lane_id_y * HEAD_DIM +
          mem_lane_id_x * 2 + k * TC_SIZE);
      *(__half2 *)(smem_k[k % 2] + warp_id * TC_SIZE * TC_SIZE +
                   i * WARP_SIZE * 2 + mem_lane_id_y * TC_SIZE +
                   mem_lane_id_x * 2) = h2;
    }

    for (k = 1; k < NUM_HEAD_WARPS; k++) {
      __syncthreads();
      for (int i = 0; i < stride_warp; i++) {
        h2 = *reinterpret_cast<const __half2 *>(
            query_ptr + warp_id * TC_SIZE * HEAD_DIM +
            i * 2 * WARP_SIZE * HEAD_DIM / TC_SIZE + mem_lane_id_y * HEAD_DIM +
            mem_lane_id_x * 2 + k * TC_SIZE);
        *(__half2 *)(smem_q[k % 2] + warp_id * TC_SIZE * TC_SIZE +
                     i * WARP_SIZE * 2 + mem_lane_id_y * TC_SIZE +
                     mem_lane_id_x * 2) = h2;

        h2 = *reinterpret_cast<const __half2 *>(
            key_ptr + warp_id * TC_SIZE * HEAD_DIM +
            i * 2 * WARP_SIZE * HEAD_DIM / TC_SIZE + mem_lane_id_y * HEAD_DIM +
            mem_lane_id_x * 2 + k * TC_SIZE);
        *(__half2 *)(smem_k[k % 2] + warp_id * TC_SIZE * TC_SIZE +
                     i * WARP_SIZE * 2 + mem_lane_id_y * TC_SIZE +
                     mem_lane_id_x * 2) = h2;
      }

      nvcuda::wmma::load_matrix_sync(
          q_frag, &(smem_q[(k - 1) % 2][warp_id * TC_SIZE * TC_SIZE]), TC_SIZE);
      for (int xi = 0; xi < NUM_WARPS; xi++) {
        nvcuda::wmma::load_matrix_sync(
            k_frag, &(smem_k[(k - 1) % 2][xi * TC_SIZE * TC_SIZE]), TC_SIZE);
        nvcuda::wmma::mma_sync(kv_out_frag[xi], q_frag, k_frag,
                               kv_out_frag[xi]);
      }
    }
    __syncthreads();
    k = NUM_HEAD_WARPS - 1;
    nvcuda::wmma::load_matrix_sync(
        q_frag, &(smem_q[k % 2][warp_id * TC_SIZE * TC_SIZE]), TC_SIZE);
    for (int xi = 0; xi < NUM_WARPS; xi++) {
      nvcuda::wmma::load_matrix_sync(
          k_frag, &(smem_k[k % 2][xi * TC_SIZE * TC_SIZE]), TC_SIZE);
      nvcuda::wmma::mma_sync(kv_out_frag[xi], q_frag, k_frag, kv_out_frag[xi]);
    }

#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
#pragma unroll
      for (int tc_yi = 0; tc_yi < 2; tc_yi++) {
#pragma unroll
        for (int tc_xi = 0; tc_xi < 2; tc_xi++) {
          kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0] =
              __hmul(kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0], sqrt_d);
          kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1] =
              __hmul(kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1], sqrt_d);
          thread_max[tc_yi] =
              hmax(thread_max[tc_yi],
                   hmax(kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0],
                        kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1]));
        }
      }
    }

#pragma unroll
    for (int s = 2; s > 0; s >>= 1) {
#pragma unroll
      for (int tc_yi = 0; tc_yi < 2; tc_yi++) {
        thread_max[tc_yi] =
            hmax(thread_max[tc_yi],
                 __shfl_xor_sync(0xffffffff, thread_max[tc_yi], s, 4));
      }
    }

#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
#pragma unroll
      for (int tc_yi = 0; tc_yi < 2; tc_yi++) {
#pragma unroll
        for (int tc_xi = 0; tc_xi < 2; tc_xi++) {
          kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0] = hexp(
              kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0] - thread_max[tc_yi]);
          kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1] = hexp(
              kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1] - thread_max[tc_yi]);
          thread_sum[tc_yi] += (kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0] +
                                kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1]);
        }
      }
    }

#pragma unroll
    for (int s = 2; s > 0; s >>= 1) {
#pragma unroll
      for (int tc_yi = 0; tc_yi < 2; tc_yi++) {
        thread_sum[tc_yi] +=
            __shfl_xor_sync(0xffffffff, thread_sum[tc_yi], s, 4);
      }
    }

    __syncthreads();
#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
      nvcuda::wmma::store_matrix_sync(
          qk_buffer_fh + warp_id * TC_SIZE * TC_SIZE, kv_out_frag[xi], TC_SIZE,
          nvcuda::wmma::mem_row_major);
      nvcuda::wmma::load_matrix_sync(
          qk_frag[xi], qk_buffer_fh + warp_id * TC_SIZE * TC_SIZE, TC_SIZE);
    }
    __syncthreads();

#pragma unroll
    for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
      nvcuda::wmma::fill_fragment(kv_out_frag[xi], 0.f);
    }

    stride_warp = TC_SIZE / warp_num;
    const __half *value_ptr = value + batch * KV_LEN * HEAD_DIM +
                              (kv_start + stride_warp * warp_id) * HEAD_DIM;
    k = 0;
    for (int i = 0; i < stride_warp * HEAD_DIM / 2 / WARP_SIZE; i++) {
      h2 = *reinterpret_cast<const __half2 *>(
          value_ptr + k * TC_SIZE * HEAD_DIM + i * WARP_SIZE * 2 + lane_id * 2);
      *(__half2 *)(smem_v[k % 2] + warp_id * stride_warp * HEAD_DIM +
                   i * WARP_SIZE * 2 + lane_id * 2) = h2;
    }

    for (k = 1; k < NUM_WARPS; k++) {
      __syncthreads();
      for (int i = 0; i < stride_warp * HEAD_DIM / 2 / WARP_SIZE; i++) {
        h2 = *reinterpret_cast<const __half2 *>(
            value_ptr + k * TC_SIZE * HEAD_DIM + i * WARP_SIZE * 2 +
            lane_id * 2);
        *(__half2 *)(smem_v[k % 2] + warp_id * stride_warp * HEAD_DIM +
                     i * WARP_SIZE * 2 + lane_id * 2) = h2;
      }

      for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
        nvcuda::wmma::load_matrix_sync(
            v_frag, &(smem_v[(k - 1) % 2][xi * TC_SIZE]), HEAD_DIM);
        nvcuda::wmma::mma_sync(kv_out_frag[xi], qk_frag[k - 1], v_frag,
                               kv_out_frag[xi]);
      }
    }
    __syncthreads();
    k = NUM_WARPS - 1;
    for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
      nvcuda::wmma::load_matrix_sync(v_frag, &(smem_v[k % 2][xi * TC_SIZE]),
                                     HEAD_DIM);
      nvcuda::wmma::mma_sync(kv_out_frag[xi], qk_frag[k], v_frag,
                             kv_out_frag[xi]);
    }

#pragma unroll
    for (int tc_yi = 0; tc_yi < 2; tc_yi++) {
      __half thread_max_new = hmax(thread_max_old[tc_yi], thread_max[tc_yi]);
      __half exp_max_old = hexp(thread_max_old[tc_yi] - thread_max_new);
      __half exp_max = hexp(thread_max[tc_yi] - thread_max_new);
      __half thread_sum_new =
          exp_max_old * thread_sum_old[tc_yi] + exp_max * thread_sum[tc_yi];
#pragma unroll
      for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
#pragma unroll
        for (int tc_xi = 0; tc_xi < 2; tc_xi++) {
          out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0] = __hdiv(
              thread_sum_old[tc_yi] * exp_max_old *
                      out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0] +
                  exp_max * (kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 0]),
              thread_sum_new);
          out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1] = __hdiv(
              thread_sum_old[tc_yi] * exp_max_old *
                      out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1] +
                  exp_max * (kv_out_frag[xi].x[tc_xi * 4 + tc_yi * 2 + 1]),
              thread_sum_new);
        }
      }
      thread_sum_old[tc_yi] = thread_sum_new;
      thread_max_old[tc_yi] = thread_max_new;
    }
  }

  for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
    nvcuda::wmma::store_matrix_sync(
        output_ptr + warp_id * TC_SIZE * HEAD_DIM + xi * TC_SIZE, out_frag[xi],
        HEAD_DIM, nvcuda::wmma::mem_row_major);
  }
}

template <int HEAD_DIM, int BASE_SEQ_LEN>
__global__ void FMHAInferKernel(const __half2 *__restrict__ query,
                                const __half2 *__restrict__ key,
                                const __half2 *__restrict__ value,
                                __half2 sqrt_d, __half2 *output,
                                const int KV_LEN) {
  static_assert(BASE_SEQ_LEN % TC_SIZE == 0 && HEAD_DIM % TC_SIZE == 0 &&
                    BASE_SEQ_LEN >= HEAD_DIM,
                "");
  const int NUM_WARPS = BASE_SEQ_LEN / TC_SIZE;
  const int NUM_HEAD_WARPS = HEAD_DIM / TC_SIZE;

  __shared__ __half2 share_buffer[BASE_SEQ_LEN * TC_SIZE * 2];
  auto qk_buffer_fh =
      reinterpret_cast<__half *>(share_buffer + BASE_SEQ_LEN * TC_SIZE);

  const unsigned int batch = blockIdx.y;
  const unsigned int Q_LEN = gridDim.x * BASE_SEQ_LEN;
  const unsigned int q_start = blockIdx.x * BASE_SEQ_LEN;
  const unsigned int tid = threadIdx.x;
  const unsigned int warp_id = tid / WARP_SIZE;
  const unsigned int lane_id = tid % WARP_SIZE;
  const unsigned int warp_num = blockDim.x / WARP_SIZE;

  const unsigned int mem_lane_id_x = lane_id % 8;
  const unsigned int mem_lane_id_y = lane_id / 8;

  __half2 *smem_q[2];
  __half2 *smem_k[2];
  __half2 *smem_v[2];

  smem_q[0] = share_buffer;
  smem_q[1] = share_buffer + 1 * BASE_SEQ_LEN * TC_SIZE / 2;
  smem_k[0] = share_buffer + 2 * BASE_SEQ_LEN * TC_SIZE / 2;
  smem_k[1] = share_buffer + 3 * BASE_SEQ_LEN * TC_SIZE / 2;
  smem_v[0] = share_buffer + 2 * BASE_SEQ_LEN * TC_SIZE / 2;
  smem_v[1] = share_buffer + 3 * BASE_SEQ_LEN * TC_SIZE / 2;

  const __half2 *query_ptr =
      query + batch * Q_LEN * HEAD_DIM / 2 + q_start * HEAD_DIM / 2;
  __half2 *output_ptr =
      output + batch * Q_LEN * HEAD_DIM / 2 + q_start * HEAD_DIM / 2;

  __half2 thread_max_old =
      __half2half2(-cuda::std::numeric_limits<__half>::infinity());
  __half2 thread_sum_old = __float2half2_rn(0.f);

  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half, nvcuda::wmma::row_major>
      q_frag;
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half, nvcuda::wmma::col_major>
      k_frag;
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half, nvcuda::wmma::row_major>
      qk_frag[NUM_WARPS];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half, nvcuda::wmma::row_major>
      v_frag;
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half>
      kv_out_frag[NUM_WARPS > NUM_HEAD_WARPS ? NUM_WARPS : NUM_HEAD_WARPS];
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half>
      out_frag[NUM_HEAD_WARPS];

#pragma unroll
  for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
    nvcuda::wmma::fill_fragment(out_frag[xi], 0.f);
  }

#pragma unroll
  for (int kv_start = 0; kv_start < KV_LEN; kv_start += BASE_SEQ_LEN) {
    const __half2 *key_ptr =
        key + batch * KV_LEN * HEAD_DIM / 2 + kv_start * HEAD_DIM / 2;

    __half2 thread_max =
        __half2half2(-cuda::std::numeric_limits<__half>::infinity());
    __half2 thread_sum = __float2half2_rn(0.f);

#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
      nvcuda::wmma::fill_fragment(kv_out_frag[xi], 0.f);
    }

    int k = 0, stride_warp = TC_SIZE * TC_SIZE / 2 / WARP_SIZE;
    __half2 h2;
    for (int i = 0; i < stride_warp; i++) {
      h2 = *(query_ptr + warp_id * TC_SIZE * HEAD_DIM / 2 +
             i * WARP_SIZE * HEAD_DIM / TC_SIZE + mem_lane_id_y * HEAD_DIM / 2 +
             mem_lane_id_x + k * TC_SIZE / 2);
      *(smem_q[k % 2] + warp_id * TC_SIZE * TC_SIZE / 2 + i * WARP_SIZE +
        mem_lane_id_y * TC_SIZE / 2 + mem_lane_id_x) = h2;

      h2 = *(key_ptr + warp_id * TC_SIZE * HEAD_DIM / 2 +
             i * WARP_SIZE * HEAD_DIM / TC_SIZE + mem_lane_id_y * HEAD_DIM / 2 +
             mem_lane_id_x + k * TC_SIZE / 2);
      *(smem_k[k % 2] + warp_id * TC_SIZE * TC_SIZE / 2 + i * WARP_SIZE +
        mem_lane_id_y * TC_SIZE / 2 + mem_lane_id_x) = h2;
    }

    for (k = 1; k < NUM_HEAD_WARPS; k++) {
      __syncthreads();
      for (int i = 0; i < stride_warp; i++) {
        h2 = *(query_ptr + warp_id * TC_SIZE * HEAD_DIM / 2 +
               i * WARP_SIZE * HEAD_DIM / TC_SIZE +
               mem_lane_id_y * HEAD_DIM / 2 + mem_lane_id_x + k * TC_SIZE / 2);
        *(smem_q[k % 2] + warp_id * TC_SIZE * TC_SIZE / 2 + i * WARP_SIZE +
          mem_lane_id_y * TC_SIZE / 2 + mem_lane_id_x) = h2;

        h2 = *(key_ptr + warp_id * TC_SIZE * HEAD_DIM / 2 +
               i * WARP_SIZE * HEAD_DIM / TC_SIZE +
               mem_lane_id_y * HEAD_DIM / 2 + mem_lane_id_x + k * TC_SIZE / 2);
        *(smem_k[k % 2] + warp_id * TC_SIZE * TC_SIZE / 2 + i * WARP_SIZE +
          mem_lane_id_y * TC_SIZE / 2 + mem_lane_id_x) = h2;
      }

      nvcuda::wmma::load_matrix_sync(
          q_frag,
          reinterpret_cast<__half *>(
              &(smem_q[(k - 1) % 2][warp_id * TC_SIZE * TC_SIZE / 2])),
          TC_SIZE);
      for (int xi = 0; xi < NUM_WARPS; xi++) {
        nvcuda::wmma::load_matrix_sync(
            k_frag,
            reinterpret_cast<__half *>(
                &(smem_k[(k - 1) % 2][xi * TC_SIZE * TC_SIZE / 2])),
            TC_SIZE);
        nvcuda::wmma::mma_sync(kv_out_frag[xi], q_frag, k_frag,
                               kv_out_frag[xi]);
      }
    }
    __syncthreads();
    k = NUM_HEAD_WARPS - 1;
    nvcuda::wmma::load_matrix_sync(
        q_frag,
        reinterpret_cast<__half *>(
            &(smem_q[k % 2][warp_id * TC_SIZE * TC_SIZE / 2])),
        TC_SIZE);
    for (int xi = 0; xi < NUM_WARPS; xi++) {
      nvcuda::wmma::load_matrix_sync(
          k_frag,
          reinterpret_cast<__half *>(
              &(smem_k[k % 2][xi * TC_SIZE * TC_SIZE / 2])),
          TC_SIZE);
      nvcuda::wmma::mma_sync(kv_out_frag[xi], q_frag, k_frag, kv_out_frag[xi]);
    }

#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
#pragma unroll
      for (int tc_xi = 0; tc_xi < 2; tc_xi++) {
        __half2 temp1 = __halves2half2(kv_out_frag[xi].x[tc_xi * 4],
                                       kv_out_frag[xi].x[tc_xi * 4 + 2]);
        __half2 temp2 = __halves2half2(kv_out_frag[xi].x[tc_xi * 4 + 1],
                                       kv_out_frag[xi].x[tc_xi * 4 + 3]);
        temp1 = __hmul2(temp1, sqrt_d);
        temp2 = __hmul2(temp2, sqrt_d);

        kv_out_frag[xi].x[tc_xi * 4] = __low2half(temp1);
        kv_out_frag[xi].x[tc_xi * 4 + 2] = __high2half(temp1);
        kv_out_frag[xi].x[tc_xi * 4 + 1] = __low2half(temp2);
        kv_out_frag[xi].x[tc_xi * 4 + 3] = __high2half(temp2);
        thread_max = hmax(thread_max, hmax(temp1, temp2));
      }
    }

#pragma unroll
    for (int s = 2; s > 0; s >>= 1) {
      thread_max =
          hmax(thread_max, __shfl_xor_sync(0xffffffff, thread_max, s, 4));
    }

#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
#pragma unroll
      for (int tc_xi = 0; tc_xi < 2; tc_xi++) {
        __half2 temp1 = __halves2half2(kv_out_frag[xi].x[tc_xi * 4],
                                       kv_out_frag[xi].x[tc_xi * 4 + 2]);
        __half2 temp2 = __halves2half2(kv_out_frag[xi].x[tc_xi * 4 + 1],
                                       kv_out_frag[xi].x[tc_xi * 4 + 3]);
        temp1 = h2exp(__hsub2(temp1, thread_max));
        temp2 = h2exp(__hsub2(temp2, thread_max));
        kv_out_frag[xi].x[tc_xi * 4] = __low2half(temp1);
        kv_out_frag[xi].x[tc_xi * 4 + 2] = __high2half(temp1);
        kv_out_frag[xi].x[tc_xi * 4 + 1] = __low2half(temp2);
        kv_out_frag[xi].x[tc_xi * 4 + 3] = __high2half(temp2);
        thread_sum = __hadd2(thread_sum, __hadd2(temp1, temp2));
      }
    }

#pragma unroll
    for (int s = 2; s > 0; s >>= 1) {
      thread_sum =
          __hadd2(thread_sum, __shfl_xor_sync(0xffffffff, thread_sum, s, 4));
    }

    __syncthreads();
#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
      nvcuda::wmma::store_matrix_sync(
          qk_buffer_fh + warp_id * TC_SIZE * TC_SIZE, kv_out_frag[xi], TC_SIZE,
          nvcuda::wmma::mem_row_major);
      nvcuda::wmma::load_matrix_sync(
          qk_frag[xi], qk_buffer_fh + warp_id * TC_SIZE * TC_SIZE, TC_SIZE);
    }
    __syncthreads();

#pragma unroll
    for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
      nvcuda::wmma::fill_fragment(kv_out_frag[xi], 0.f);
    }

    stride_warp = TC_SIZE / warp_num;
    const __half2 *value_ptr =
        value + batch * KV_LEN * HEAD_DIM / 2 +
        (kv_start + stride_warp * warp_id) * HEAD_DIM / 2;
    k = 0;
    for (int i = 0; i < stride_warp * HEAD_DIM / 2 / WARP_SIZE; i++) {
      h2 = *(value_ptr + k * TC_SIZE * HEAD_DIM / 2 + i * WARP_SIZE + lane_id);
      *(smem_v[k % 2] + warp_id * stride_warp * HEAD_DIM / 2 + i * WARP_SIZE +
        lane_id) = h2;
    }

    for (k = 1; k < NUM_WARPS; k++) {
      __syncthreads();
      for (int i = 0; i < stride_warp * HEAD_DIM / 2 / WARP_SIZE; i++) {
        h2 =
            *(value_ptr + k * TC_SIZE * HEAD_DIM / 2 + i * WARP_SIZE + lane_id);
        *(smem_v[k % 2] + warp_id * stride_warp * HEAD_DIM / 2 + i * WARP_SIZE +
          lane_id) = h2;
      }

      for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
        nvcuda::wmma::load_matrix_sync(
            v_frag,
            reinterpret_cast<__half *>(
                &(smem_v[(k - 1) % 2][xi * TC_SIZE / 2])),
            HEAD_DIM);
        nvcuda::wmma::mma_sync(kv_out_frag[xi], qk_frag[k - 1], v_frag,
                               kv_out_frag[xi]);
      }
    }
    __syncthreads();
    k = NUM_WARPS - 1;
    for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
      nvcuda::wmma::load_matrix_sync(
          v_frag,
          reinterpret_cast<__half *>(&(smem_v[k % 2][xi * TC_SIZE / 2])),
          HEAD_DIM);
      nvcuda::wmma::mma_sync(kv_out_frag[xi], qk_frag[k], v_frag,
                             kv_out_frag[xi]);
    }

    __half2 thread_max_new = hmax(thread_max_old, thread_max);
    __half2 exp_max_old = h2exp(__hsub2(thread_max_old, thread_max_new));
    __half2 exp_max = h2exp(__hsub2(thread_max, thread_max_new));
    __half2 thread_sum_new = __hadd2(__hmul2(exp_max_old, thread_sum_old),
                                     __hmul2(exp_max, thread_sum));
#pragma unroll
    for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
#pragma unroll
      for (int tc_xi = 0; tc_xi < 2; tc_xi++) {
        __half2 temp1 = __halves2half2(out_frag[xi].x[tc_xi * 4],
                                       out_frag[xi].x[tc_xi * 4 + 2]);
        __half2 temp2 = __halves2half2(out_frag[xi].x[tc_xi * 4 + 1],
                                       out_frag[xi].x[tc_xi * 4 + 3]);
        __half2 temp1_kv = __halves2half2(kv_out_frag[xi].x[tc_xi * 4],
                                          kv_out_frag[xi].x[tc_xi * 4 + 2]);
        __half2 temp2_kv = __halves2half2(kv_out_frag[xi].x[tc_xi * 4 + 1],
                                          kv_out_frag[xi].x[tc_xi * 4 + 3]);

        temp1 = __h2div(
            __hfma2(exp_max, temp1_kv,
                    __hmul2(__hmul2(thread_sum_old, exp_max_old), temp1)),
            thread_sum_new);
        temp2 = __h2div(
            __hfma2(exp_max, temp2_kv,
                    __hmul2(__hmul2(thread_sum_old, exp_max_old), temp2)),
            thread_sum_new);

        out_frag[xi].x[tc_xi * 4] = __low2half(temp1);
        out_frag[xi].x[tc_xi * 4 + 2] = __high2half(temp1);
        out_frag[xi].x[tc_xi * 4 + 1] = __low2half(temp2);
        out_frag[xi].x[tc_xi * 4 + 3] = __high2half(temp2);
      }
    }
    thread_sum_old = thread_sum_new;
    thread_max_old = thread_max_new;
  }

  for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
    nvcuda::wmma::store_matrix_sync(
        reinterpret_cast<__half *>(
            output_ptr + warp_id * TC_SIZE * HEAD_DIM / 2 + xi * TC_SIZE / 2),
        out_frag[xi], HEAD_DIM, nvcuda::wmma::mem_row_major);
  }
}

template <typename T> __forceinline__ __device__ T sign_05(T x) {
  if (x > 0) {
    return 0.5f;
  }
  return -0.5f;
}

template <typename T> __forceinline__ __device__ int8_t T2int8(T a) {
  a = a > 127 ? 127 : a;
  a = a < -128 ? -128 : a;
  return int8_t(a + sign_05<T>(a));
}

template <> __forceinline__ __device__ int8_t T2int8(__half a) {
  a = __hgt(a, __int2half_rn(127)) ? __int2half_rn(127) : a;
  a = __hlt(a, __int2half_rn(-128)) ? __int2half_rn(-128) : a;
  return int8_t(__half2int_rn(a));
}

template <typename T> __forceinline__ __device__ uint8_t T2uint8(T a) {
  a = a > 255 ? 255 : a;
  a = a < 0 ? 0 : a;
  return uint8_t(a + 0.5);
}

template <> __forceinline__ __device__ uint8_t T2uint8(__half a) {
  a = __hgt(a, __int2half_rn(255)) ? __int2half_rn(255) : a;
  a = __hlt(a, __int2half_rn(0)) ? __int2half_rn(0) : a;
  return uint8_t(__half2int_rn(a));
}

#if __CUDA_ARCH__ >= 800
template <int HEAD_DIM, int BASE_SEQ_LEN>
__global__ void
FMHAInferInt8Kernel(const int8_4 *__restrict__ query, const float scale_q,
                    const int8_4 *__restrict__ key, const float scale_k,
                    const int8_4 *__restrict__ value, const float scale_v,
                    const float sqrt_d, int8_4 *output, const float scale_o,
                    const int KV_LEN) {
  static_assert(BASE_SEQ_LEN % TC_SIZE == 0 && HEAD_DIM % TC_SIZE == 0 &&
                    BASE_SEQ_LEN >= HEAD_DIM,
                "");
  constexpr int NUM_WARPS = BASE_SEQ_LEN / TC_SIZE;
  constexpr int NUM_HEAD_WARPS = HEAD_DIM / TC_SIZE;
  constexpr int NUM_WARPS_MAX =
      NUM_WARPS > NUM_HEAD_WARPS ? NUM_WARPS : NUM_HEAD_WARPS;

  const unsigned int batch = blockIdx.y;
  const unsigned int Q_LEN = gridDim.x * BASE_SEQ_LEN;
  const unsigned int q_start = blockIdx.x * BASE_SEQ_LEN;
  const unsigned int tid = threadIdx.x;
  const unsigned int warp_id = tid / WARP_SIZE;
  const unsigned int lane_id = tid % WARP_SIZE;

  const unsigned int mem_lane_id_x = lane_id % 4;
  const unsigned int mem_lane_id_y = lane_id / 4;

  const float scale_qkv = scale_v * __frcp_rn(255.f);
  const __half2 scale_softmax_re = __float2half2_rn(255.f);
  const __half2 scale_o_re = __float2half2_rn(__frcp_rn(scale_o));

  int8_4 query_ma[NUM_HEAD_WARPS * 2];
  int8_4 key_mb[2];
  uint8_t softmax_ma[NUM_WARPS * 8];
  int8_4 value_mb[NUM_WARPS * 2];
  __half out_h[NUM_HEAD_WARPS * 8] = {0.f};
  auto out_h2 = reinterpret_cast<__half2 *>(out_h);
  auto out_i82 = reinterpret_cast<int8_2 *>(out_h);

  __half2 thread_max_old =
      __half2half2(-cuda::std::numeric_limits<__half>::infinity());
  __half2 thread_sum_old = __float2half2_rn(0.f);

  const int8_4 *query_ptr =
      query + (batch * Q_LEN + q_start + warp_id * TC_SIZE) * HEAD_DIM / 4;
  auto output_ptr = reinterpret_cast<int8_2 *>(
      output + (batch * Q_LEN + q_start + warp_id * TC_SIZE) * HEAD_DIM / 4);

#pragma unroll
  for (int head_id = 0; head_id < NUM_HEAD_WARPS; head_id++) {
    query_ma[head_id * 2 + 0] = *(query_ptr + head_id * TC_SIZE / 4 +
                                  mem_lane_id_y * HEAD_DIM / 4 + mem_lane_id_x);
    query_ma[head_id * 2 + 1] =
        *(query_ptr + head_id * TC_SIZE / 4 + TC_SIZE * HEAD_DIM / 8 +
          mem_lane_id_y * HEAD_DIM / 4 + mem_lane_id_x);
  }
#pragma unroll
  for (int kv_start = 0; kv_start < KV_LEN; kv_start += BASE_SEQ_LEN) {
    const int8_4 *key_ptr = key + (batch * KV_LEN + kv_start) * HEAD_DIM / 4;

    int32_t mc[NUM_WARPS_MAX * 8] = {0};
    auto mc_f = reinterpret_cast<float *>(mc);
    auto mc_f2 = reinterpret_cast<float2 *>(mc);
    auto mc_h = reinterpret_cast<__half *>(mc);
    auto mc_h2 = reinterpret_cast<__half2 *>(mc);

    __half2 thread_max =
        __half2half2(-cuda::std::numeric_limits<__half>::infinity());
    __half2 thread_sum = __float2half2_rn(0.f);

#pragma unroll
    for (int head_id = 0; head_id < NUM_HEAD_WARPS; head_id += 2) {
#pragma unroll
      for (int k_i = 0; k_i < NUM_WARPS; k_i++) {
#pragma unroll
        for (int j = 0; j < 2; j++) {
          key_mb[0] = *(key_ptr + k_i * HEAD_DIM * TC_SIZE / 4 +
                        head_id * TC_SIZE / 4 + j * TC_SIZE * HEAD_DIM / 8 +
                        mem_lane_id_y * HEAD_DIM / 4 + mem_lane_id_x);
          key_mb[1] =
              *(key_ptr + k_i * HEAD_DIM * TC_SIZE / 4 +
                (head_id + 1) * TC_SIZE / 4 + j * TC_SIZE * HEAD_DIM / 8 +
                mem_lane_id_y * HEAD_DIM / 4 + mem_lane_id_x);
          asm("mma.sync.aligned.m16n8k32.row.col.s32.s8.s8.s32 "
              " { %0, %1, %2, %3 }, "
              " { %4, %5, %6, %7 }, "
              " { %8, %9 }, "
              " { %10, %11, %12, %13 }; "
              : "=r"(mc[k_i * 8 + j * 2 + 0]), "=r"(mc[k_i * 8 + j * 2 + 1]),
                "=r"(mc[k_i * 8 + j * 2 + 4]), "=r"(mc[k_i * 8 + j * 2 + 5])
              : "r"(*reinterpret_cast<uint32_t *>(&query_ma[head_id * 2 + 0])),
                "r"(*reinterpret_cast<uint32_t *>(&query_ma[head_id * 2 + 1])),
                "r"(*reinterpret_cast<uint32_t *>(&query_ma[head_id * 2 + 2])),
                "r"(*reinterpret_cast<uint32_t *>(&query_ma[head_id * 2 + 3])),
                "r"(*reinterpret_cast<uint32_t *>(&key_mb[0])),
                "r"(*reinterpret_cast<uint32_t *>(&key_mb[1])),
                "r"(mc[k_i * 8 + j * 2 + 0]), "r"(mc[k_i * 8 + j * 2 + 1]),
                "r"(mc[k_i * 8 + j * 2 + 4]), "r"(mc[k_i * 8 + j * 2 + 5]));
        }
      }
    }

#pragma unroll
    for (int i = 0; i < NUM_WARPS; i++) {
#pragma unroll
      for (int j = 0; j < 8; j++) {
        mc_f[i * 8 + j] = mc[i * 8 + j] * sqrt_d;
      }
#pragma unroll
      for (int j = 0; j < 4; j++) {
        mc_h2[i * 4 + j] = __float22half2_rn(mc_f2[i * 4 + j]);
      }

      thread_max =
          hmax(thread_max,
               hmax(hmax(__halves2half2(mc_h[i * 8 + 0], mc_h[i * 8 + 4]),
                         __halves2half2(mc_h[i * 8 + 1], mc_h[i * 8 + 5])),
                    hmax(__halves2half2(mc_h[i * 8 + 2], mc_h[i * 8 + 6]),
                         __halves2half2(mc_h[i * 8 + 3], mc_h[i * 8 + 7]))));
    }

#pragma unroll
    for (int s = 2; s > 0; s >>= 1) {
      thread_max =
          hmax(thread_max, __shfl_xor_sync(0xffffffff, thread_max, s, 4));
    }

#pragma unroll
    for (int i = 0; i < NUM_WARPS; i++) {
#pragma unroll
      for (int j = 0; j < 2; j++) {
        mc_h2[i * 4 + j] =
            h2exp(__hsub2(mc_h2[i * 4 + j], __low2half2(thread_max)));
        mc_h2[i * 4 + j + 2] =
            h2exp(__hsub2(mc_h2[i * 4 + j + 2], __high2half2(thread_max)));

        thread_sum =
            __hadd2(thread_sum,
                    __hadd2(__halves2half2(__high2half(mc_h2[i * 4 + j]),
                                           __high2half(mc_h2[i * 4 + j + 2])),
                            __halves2half2(__low2half(mc_h2[i * 4 + j]),
                                           __low2half(mc_h2[i * 4 + j + 2]))));

        mc_h2[i * 4 + j] = __hmul2(mc_h2[i * 4 + j], scale_softmax_re);
        mc_h2[i * 4 + j + 2] = __hmul2(mc_h2[i * 4 + j + 2], scale_softmax_re);
        softmax_ma[i * 8 + j * 2 + 0] = T2uint8(__low2half(mc_h2[i * 4 + j]));
        softmax_ma[i * 8 + j * 2 + 1] = T2uint8(__high2half(mc_h2[i * 4 + j]));
        softmax_ma[i * 8 + j * 2 + 4] =
            T2uint8(__low2half(mc_h2[i * 4 + j + 2]));
        softmax_ma[i * 8 + j * 2 + 5] =
            T2uint8(__high2half(mc_h2[i * 4 + j + 2]));
      }
    }

#pragma unroll
    for (int s = 2; s > 0; s >>= 1) {
      thread_sum =
          __hadd2(thread_sum, __shfl_xor_sync(0xffffffff, thread_sum, s, 4));
    }

#pragma unroll
    for (int i = 0; i < NUM_HEAD_WARPS; i++) {
#pragma unroll
      for (int j = 0; j < 8; j++) {
        mc[i * 8 + j] = 0;
      }
    }

    int8_4 *val = key_mb;
    const int8_t *value_ptr = reinterpret_cast<const int8_t *>(value) +
                              batch * KV_LEN * HEAD_DIM + kv_start * HEAD_DIM +
                              mem_lane_id_y + mem_lane_id_x * 2 * HEAD_DIM;
#pragma unroll
    for (int head_id = 0; head_id < NUM_HEAD_WARPS; head_id++) {
#pragma unroll
      for (int v_i = 0; v_i < NUM_WARPS; v_i += 2) {
        val[0].x = *(value_ptr + v_i * TC_SIZE * HEAD_DIM);
        val[0].y = *(value_ptr + v_i * TC_SIZE * HEAD_DIM + HEAD_DIM);
        val[0].z =
            *(value_ptr + v_i * TC_SIZE * HEAD_DIM + TC_SIZE * HEAD_DIM / 2);
        val[0].w = *(value_ptr + v_i * TC_SIZE * HEAD_DIM +
                     TC_SIZE * HEAD_DIM / 2 + HEAD_DIM);
        val[1].x = *(value_ptr + (v_i + 1) * TC_SIZE * HEAD_DIM);
        val[1].y = *(value_ptr + (v_i + 1) * TC_SIZE * HEAD_DIM + HEAD_DIM);
        val[1].z = *(value_ptr + (v_i + 1) * TC_SIZE * HEAD_DIM +
                     TC_SIZE * HEAD_DIM / 2);
        val[1].w = *(value_ptr + (v_i + 1) * TC_SIZE * HEAD_DIM +
                     TC_SIZE * HEAD_DIM / 2 + HEAD_DIM);

        asm("mma.sync.aligned.m16n8k32.row.col.s32.u8.s8.s32 "
            " { %0, %1, %2, %3 }, "
            " { %4, %5, %6, %7 }, "
            " { %8, %9 }, "
            " { %10, %11, %12, %13 }; "
            : "=r"(mc[head_id * 8 + 0]), "=r"(mc[head_id * 8 + 1]),
              "=r"(mc[head_id * 8 + 4]), "=r"(mc[head_id * 8 + 5])
            : "r"(*reinterpret_cast<uint32_t *>(&softmax_ma[v_i * 8 + 0])),
              "r"(*reinterpret_cast<uint32_t *>(&softmax_ma[v_i * 8 + 4])),
              "r"(*reinterpret_cast<uint32_t *>(&softmax_ma[v_i * 8 + 8])),
              "r"(*reinterpret_cast<uint32_t *>(&softmax_ma[v_i * 8 + 12])),
              "r"(*reinterpret_cast<uint32_t *>(val)),
              "r"(*reinterpret_cast<uint32_t *>(val + 1)),
              "r"(mc[head_id * 8 + 0]), "r"(mc[head_id * 8 + 1]),
              "r"(mc[head_id * 8 + 4]), "r"(mc[head_id * 8 + 5]));

        val[0].x = *(value_ptr + v_i * TC_SIZE * HEAD_DIM + TC_SIZE / 2);
        val[0].y =
            *(value_ptr + v_i * TC_SIZE * HEAD_DIM + HEAD_DIM + TC_SIZE / 2);
        val[0].z = *(value_ptr + v_i * TC_SIZE * HEAD_DIM +
                     TC_SIZE * HEAD_DIM / 2 + TC_SIZE / 2);
        val[0].w = *(value_ptr + v_i * TC_SIZE * HEAD_DIM +
                     TC_SIZE * HEAD_DIM / 2 + HEAD_DIM + TC_SIZE / 2);
        val[1].x = *(value_ptr + (v_i + 1) * TC_SIZE * HEAD_DIM + TC_SIZE / 2);
        val[1].y = *(value_ptr + (v_i + 1) * TC_SIZE * HEAD_DIM + HEAD_DIM +
                     TC_SIZE / 2);
        val[1].z = *(value_ptr + (v_i + 1) * TC_SIZE * HEAD_DIM +
                     TC_SIZE * HEAD_DIM / 2 + TC_SIZE / 2);
        val[1].w = *(value_ptr + (v_i + 1) * TC_SIZE * HEAD_DIM +
                     TC_SIZE * HEAD_DIM / 2 + HEAD_DIM + TC_SIZE / 2);

        asm("mma.sync.aligned.m16n8k32.row.col.s32.u8.s8.s32 "
            " { %0, %1, %2, %3 }, "
            " { %4, %5, %6, %7 }, "
            " { %8, %9 }, "
            " { %10, %11, %12, %13 }; "
            : "=r"(mc[head_id * 8 + 2]), "=r"(mc[head_id * 8 + 3]),
              "=r"(mc[head_id * 8 + 6]), "=r"(mc[head_id * 8 + 7])
            : "r"(*reinterpret_cast<uint32_t *>(&softmax_ma[v_i * 8 + 0])),
              "r"(*reinterpret_cast<uint32_t *>(&softmax_ma[v_i * 8 + 4])),
              "r"(*reinterpret_cast<uint32_t *>(&softmax_ma[v_i * 8 + 8])),
              "r"(*reinterpret_cast<uint32_t *>(&softmax_ma[v_i * 8 + 12])),
              "r"(*reinterpret_cast<uint32_t *>(val)),
              "r"(*reinterpret_cast<uint32_t *>(val + 1)),
              "r"(mc[head_id * 8 + 2]), "r"(mc[head_id * 8 + 3]),
              "r"(mc[head_id * 8 + 6]), "r"(mc[head_id * 8 + 7]));
      }
      value_ptr += TC_SIZE;
    }

    __half2 thread_max_new = hmax(thread_max_old, thread_max);
    __half2 exp_max_old = h2exp(__hsub2(thread_max_old, thread_max_new));
    __half2 exp_max = h2exp(__hsub2(thread_max, thread_max_new));
    __half2 thread_sum_new = __hadd2(__hmul2(exp_max_old, thread_sum_old),
                                     __hmul2(exp_max, thread_sum));
    exp_max_old = __hmul2(thread_sum_old, exp_max_old);

    __half2 exp_max_low = __low2half2(exp_max);
    __half2 exp_max_high = __high2half2(exp_max);
    __half2 exp_max_old_low = __low2half2(exp_max_old);
    __half2 exp_max_old_high = __high2half2(exp_max_old);
    __half2 thread_sum_new_low = __low2half2(thread_sum_new);
    __half2 thread_sum_new_high = __high2half2(thread_sum_new);

#pragma unroll
    for (int i = 0; i < NUM_HEAD_WARPS; i++) {
#pragma unroll
      for (int j = 0; j < 8; j++) {
        mc_f[i * 8 + j] = mc[i * 8 + j] * scale_qkv;
        if (j % 2 == 1) {
          mc_h2[i * 4 + j / 2] = __float22half2_rn(mc_f2[i * 4 + j / 2]);
        }
      }
#pragma unroll
      for (int j = 0; j < 2; j++) {
        out_h2[i * 4 + j] =
            __h2div(__hfma2(exp_max_low, mc_h2[i * 4 + j],
                            __hmul2(exp_max_old_low, out_h2[i * 4 + j])),
                    thread_sum_new_low);
        out_h2[i * 4 + j + 2] =
            __h2div(__hfma2(exp_max_high, mc_h2[i * 4 + j + 2],
                            __hmul2(exp_max_old_high, out_h2[i * 4 + j + 2])),
                    thread_sum_new_high);
      }
    }
    thread_sum_old = thread_sum_new;
    thread_max_old = thread_max_new;
  }

#pragma unroll
  for (int i = 0; i < NUM_HEAD_WARPS; i++) {
#pragma unroll
    for (int j = 0; j < 4; j++) {
      out_h2[i * 4 + j] = __hmul2(out_h2[i * 4 + j], scale_o_re);
      out_i82[i * 4 + j].x = T2int8(out_h2[i * 4 + j].x);
      out_i82[i * 4 + j].y = T2int8(out_h2[i * 4 + j].y);
    }

    output_ptr[mem_lane_id_y * HEAD_DIM / 2 + mem_lane_id_x] =
        out_i82[i * 4 + 0];
    output_ptr[mem_lane_id_y * HEAD_DIM / 2 + mem_lane_id_x + 4] =
        out_i82[i * 4 + 1];
    output_ptr[mem_lane_id_y * HEAD_DIM / 2 + mem_lane_id_x +
               TC_SIZE * HEAD_DIM / 4] = out_i82[i * 4 + 2];
    output_ptr[mem_lane_id_y * HEAD_DIM / 2 + mem_lane_id_x +
               TC_SIZE * HEAD_DIM / 4 + 4] = out_i82[i * 4 + 3];

    output_ptr += TC_SIZE / 2;
  }
}
#elif __CUDA_ARCH__ >= 750
template <int HEAD_DIM, int BASE_SEQ_LEN>
__global__ void
FMHAInferInt8Kernel(const int8_4 *__restrict__ query, const float scale_q,
                    const int8_4 *__restrict__ key, const float scale_k,
                    const int8_4 *__restrict__ value, const float scale_v,
                    const float sqrt_d, int8_4 *output, const float scale_o,
                    const int KV_LEN) {
  static_assert(BASE_SEQ_LEN % TC_SIZE == 0 && HEAD_DIM % TC_SIZE == 0 &&
                    BASE_SEQ_LEN >= HEAD_DIM,
                "");
  constexpr int NUM_WARPS = BASE_SEQ_LEN / TC_SIZE;
  constexpr int NUM_HEAD_WARPS = HEAD_DIM / TC_SIZE;
  constexpr int NUM_WARPS_MAX =
      NUM_WARPS > NUM_HEAD_WARPS ? NUM_WARPS : NUM_HEAD_WARPS;

  const unsigned int batch = blockIdx.y;
  const unsigned int Q_LEN = gridDim.x * BASE_SEQ_LEN;
  const unsigned int q_start = blockIdx.x * BASE_SEQ_LEN;
  const unsigned int tid = threadIdx.x;
  const unsigned int warp_id = tid / WARP_SIZE;
  const unsigned int lane_id = tid % WARP_SIZE;

  const unsigned int mem_lane_id_x = lane_id % 4;
  const unsigned int mem_lane_id_y = lane_id / 4;

  const float scale_qkv = scale_v * __frcp_rn(255.f);
  const __half2 scale_softmax_re = __float2half2_rn(255.f);
  const __half2 scale_o_re = __float2half2_rn(__frcp_rn(scale_o));

  int8_4 query_ma[NUM_HEAD_WARPS * 2];
  int8_4 key_mb[2];
  uint8_t softmax_ma[NUM_WARPS * 8];
  int8_4 value_mb[NUM_WARPS * 2];
  __half out_h[NUM_HEAD_WARPS * 8] = {0.f};
  auto out_h2 = reinterpret_cast<__half2 *>(out_h);
  auto out_i82 = reinterpret_cast<int8_2 *>(out_h);

  __half2 thread_max_old =
      __half2half2(-cuda::std::numeric_limits<__half>::infinity());
  __half2 thread_sum_old = __float2half2_rn(0.f);

  const int8_4 *query_ptr =
      query + (batch * Q_LEN + q_start + warp_id * TC_SIZE) * HEAD_DIM / 4;
  auto output_ptr = reinterpret_cast<int8_2 *>(
      output + (batch * Q_LEN + q_start + warp_id * TC_SIZE) * HEAD_DIM / 4);

#pragma unroll
  for (int head_id = 0; head_id < NUM_HEAD_WARPS; head_id++) {
    query_ma[head_id * 2 + 0] = *(query_ptr + head_id * TC_SIZE / 4 +
                                  mem_lane_id_y * HEAD_DIM / 4 + mem_lane_id_x);
    query_ma[head_id * 2 + 1] =
        *(query_ptr + head_id * TC_SIZE / 4 + TC_SIZE * HEAD_DIM / 8 +
          mem_lane_id_y * HEAD_DIM / 4 + mem_lane_id_x);
  }
#pragma unroll
  for (int kv_start = 0; kv_start < KV_LEN; kv_start += BASE_SEQ_LEN) {
    const int8_4 *key_ptr = key + (batch * KV_LEN + kv_start) * HEAD_DIM / 4;

    int32_t mc[NUM_WARPS_MAX * 8] = {0};
    auto mc_f = reinterpret_cast<float *>(mc);
    auto mc_f2 = reinterpret_cast<float2 *>(mc);
    auto mc_h = reinterpret_cast<__half *>(mc);
    auto mc_h2 = reinterpret_cast<__half2 *>(mc);

    __half2 thread_max =
        __half2half2(-cuda::std::numeric_limits<__half>::infinity());
    __half2 thread_sum = __float2half2_rn(0.f);

#pragma unroll
    for (int head_id = 0; head_id < NUM_HEAD_WARPS; head_id++) {
#pragma unroll
      for (int k_i = 0; k_i < NUM_WARPS; k_i++) {
        key_mb[0] =
            *(key_ptr + k_i * HEAD_DIM * TC_SIZE / 4 + head_id * TC_SIZE / 4 +
              mem_lane_id_y * HEAD_DIM / 4 + mem_lane_id_x);
        key_mb[1] = *(key_ptr + k_i * HEAD_DIM * TC_SIZE / 4 +
                      head_id * TC_SIZE / 4 + TC_SIZE * HEAD_DIM / 8 +
                      mem_lane_id_y * HEAD_DIM / 4 + mem_lane_id_x);
#pragma unroll
        for (int i = 0; i < 2; i++) {
#pragma unroll
          for (int j = 0; j < 2; j++) {
            asm("mma.sync.aligned.m8n8k16.row.col.s32.s8.s8.s32 "
                " { %0, %1 }, "
                " { %2 }, "
                " { %3 }, "
                " { %4, %5 }; "
                : "=r"(mc[k_i * 8 + i * 4 + j * 2 + 0]),
                  "=r"(mc[k_i * 8 + i * 4 + j * 2 + 1])
                : "r"(*reinterpret_cast<uint32_t *>(
                      &query_ma[head_id * 2 + i])),
                  "r"(*reinterpret_cast<uint32_t *>(&key_mb[j])),
                  "r"(mc[k_i * 8 + i * 4 + j * 2 + 0]),
                  "r"(mc[k_i * 8 + i * 4 + j * 2 + 1]));
          }
        }
      }
    }

#pragma unroll
    for (int i = 0; i < NUM_WARPS; i++) {
#pragma unroll
      for (int j = 0; j < 8; j++) {
        mc_f[i * 8 + j] = mc[i * 8 + j] * sqrt_d;
      }
#pragma unroll
      for (int j = 0; j < 4; j++) {
        mc_h2[i * 4 + j] = __float22half2_rn(mc_f2[i * 4 + j]);
      }

      thread_max =
          hmax(thread_max,
               hmax(hmax(__halves2half2(mc_h[i * 8 + 0], mc_h[i * 8 + 4]),
                         __halves2half2(mc_h[i * 8 + 1], mc_h[i * 8 + 5])),
                    hmax(__halves2half2(mc_h[i * 8 + 2], mc_h[i * 8 + 6]),
                         __halves2half2(mc_h[i * 8 + 3], mc_h[i * 8 + 7]))));
    }

#pragma unroll
    for (int s = 2; s > 0; s >>= 1) {
      thread_max =
          hmax(thread_max, __shfl_xor_sync(0xffffffff, thread_max, s, 4));
    }

#pragma unroll
    for (int i = 0; i < NUM_WARPS; i++) {
#pragma unroll
      for (int j = 0; j < 2; j++) {
        mc_h2[i * 4 + j] =
            h2exp(__hsub2(mc_h2[i * 4 + j], __low2half2(thread_max)));
        mc_h2[i * 4 + j + 2] =
            h2exp(__hsub2(mc_h2[i * 4 + j + 2], __high2half2(thread_max)));

        thread_sum =
            __hadd2(thread_sum,
                    __hadd2(__halves2half2(__high2half(mc_h2[i * 4 + j]),
                                           __high2half(mc_h2[i * 4 + j + 2])),
                            __halves2half2(__low2half(mc_h2[i * 4 + j]),
                                           __low2half(mc_h2[i * 4 + j + 2]))));

        mc_h2[i * 4 + j] = __hmul2(mc_h2[i * 4 + j], scale_softmax_re);
        mc_h2[i * 4 + j + 2] = __hmul2(mc_h2[i * 4 + j + 2], scale_softmax_re);
        softmax_ma[i * 8 + j * 2 + 0] = T2uint8(__low2half(mc_h2[i * 4 + j]));
        softmax_ma[i * 8 + j * 2 + 1] = T2uint8(__high2half(mc_h2[i * 4 + j]));
        softmax_ma[i * 8 + j * 2 + 4] =
            T2uint8(__low2half(mc_h2[i * 4 + j + 2]));
        softmax_ma[i * 8 + j * 2 + 5] =
            T2uint8(__high2half(mc_h2[i * 4 + j + 2]));
      }
    }

#pragma unroll
    for (int s = 2; s > 0; s >>= 1) {
      thread_sum =
          __hadd2(thread_sum, __shfl_xor_sync(0xffffffff, thread_sum, s, 4));
    }

#pragma unroll
    for (int i = 0; i < NUM_HEAD_WARPS; i++) {
#pragma unroll
      for (int j = 0; j < 8; j++) {
        mc[i * 8 + j] = 0;
      }
    }

    int8_4 val;
    const int8_t *value_ptr = reinterpret_cast<const int8_t *>(value) +
                              batch * KV_LEN * HEAD_DIM + kv_start * HEAD_DIM +
                              mem_lane_id_y + mem_lane_id_x * 2 * HEAD_DIM;
#pragma unroll
    for (int head_id = 0; head_id < NUM_HEAD_WARPS; head_id++) {
#pragma unroll
      for (int v_i = 0; v_i < NUM_WARPS; v_i++) {
        val.x = *(value_ptr + v_i * TC_SIZE * HEAD_DIM);
        val.y = *(value_ptr + v_i * TC_SIZE * HEAD_DIM + HEAD_DIM);
        val.z =
            *(value_ptr + v_i * TC_SIZE * HEAD_DIM + TC_SIZE * HEAD_DIM / 2);
        val.w = *(value_ptr + v_i * TC_SIZE * HEAD_DIM +
                  TC_SIZE * HEAD_DIM / 2 + HEAD_DIM);

        asm("mma.sync.aligned.m8n8k16.row.col.s32.u8.s8.s32 "
            " { %0, %1 }, "
            " { %2 }, "
            " { %3 }, "
            " { %4, %5 }; "
            : "=r"(mc[head_id * 8 + 0]), "=r"(mc[head_id * 8 + 1])
            : "r"(*reinterpret_cast<uint32_t *>(&softmax_ma[v_i * 8 + 0])),
              "r"(*reinterpret_cast<uint32_t *>(&val)),
              "r"(mc[head_id * 8 + 0]), "r"(mc[head_id * 8 + 1]));
        asm("mma.sync.aligned.m8n8k16.row.col.s32.u8.s8.s32 "
            " { %0, %1 }, "
            " { %2 }, "
            " { %3 }, "
            " { %4, %5 }; "
            : "=r"(mc[head_id * 8 + 4]), "=r"(mc[head_id * 8 + 5])
            : "r"(*reinterpret_cast<uint32_t *>(&softmax_ma[v_i * 8 + 4])),
              "r"(*reinterpret_cast<uint32_t *>(&val)),
              "r"(mc[head_id * 8 + 4]), "r"(mc[head_id * 8 + 5]));

        val.x = *(value_ptr + v_i * TC_SIZE * HEAD_DIM + TC_SIZE / 2);
        val.y =
            *(value_ptr + v_i * TC_SIZE * HEAD_DIM + HEAD_DIM + TC_SIZE / 2);
        val.z = *(value_ptr + v_i * TC_SIZE * HEAD_DIM +
                  TC_SIZE * HEAD_DIM / 2 + TC_SIZE / 2);
        val.w = *(value_ptr + v_i * TC_SIZE * HEAD_DIM +
                  TC_SIZE * HEAD_DIM / 2 + HEAD_DIM + TC_SIZE / 2);

        asm("mma.sync.aligned.m8n8k16.row.col.s32.u8.s8.s32 "
            " { %0, %1 }, "
            " { %2 }, "
            " { %3 }, "
            " { %4, %5 }; "
            : "=r"(mc[head_id * 8 + 2]), "=r"(mc[head_id * 8 + 3])
            : "r"(*reinterpret_cast<uint32_t *>(&softmax_ma[v_i * 8 + 0])),
              "r"(*reinterpret_cast<uint32_t *>(&val)),
              "r"(mc[head_id * 8 + 2]), "r"(mc[head_id * 8 + 3]));
        asm("mma.sync.aligned.m8n8k16.row.col.s32.u8.s8.s32 "
            " { %0, %1 }, "
            " { %2 }, "
            " { %3 }, "
            " { %4, %5 }; "
            : "=r"(mc[head_id * 8 + 6]), "=r"(mc[head_id * 8 + 7])
            : "r"(*reinterpret_cast<uint32_t *>(&softmax_ma[v_i * 8 + 4])),
              "r"(*reinterpret_cast<uint32_t *>(&val)),
              "r"(mc[head_id * 8 + 6]), "r"(mc[head_id * 8 + 7]));
      }
      value_ptr += TC_SIZE;
    }

    __half2 thread_max_new = hmax(thread_max_old, thread_max);
    __half2 exp_max_old = h2exp(__hsub2(thread_max_old, thread_max_new));
    __half2 exp_max = h2exp(__hsub2(thread_max, thread_max_new));
    __half2 thread_sum_new = __hadd2(__hmul2(exp_max_old, thread_sum_old),
                                     __hmul2(exp_max, thread_sum));
    exp_max_old = __hmul2(thread_sum_old, exp_max_old);

    __half2 exp_max_low = __low2half2(exp_max);
    __half2 exp_max_high = __high2half2(exp_max);
    __half2 exp_max_old_low = __low2half2(exp_max_old);
    __half2 exp_max_old_high = __high2half2(exp_max_old);
    __half2 thread_sum_new_low = __low2half2(thread_sum_new);
    __half2 thread_sum_new_high = __high2half2(thread_sum_new);

#pragma unroll
    for (int i = 0; i < NUM_HEAD_WARPS; i++) {
#pragma unroll
      for (int j = 0; j < 8; j++) {
        mc_f[i * 8 + j] = mc[i * 8 + j] * scale_qkv;
        if (j % 2 == 1) {
          mc_h2[i * 4 + j / 2] = __float22half2_rn(mc_f2[i * 4 + j / 2]);
        }
      }
#pragma unroll
      for (int j = 0; j < 2; j++) {
        out_h2[i * 4 + j] =
            __h2div(__hfma2(exp_max_low, mc_h2[i * 4 + j],
                            __hmul2(exp_max_old_low, out_h2[i * 4 + j])),
                    thread_sum_new_low);
        out_h2[i * 4 + j + 2] =
            __h2div(__hfma2(exp_max_high, mc_h2[i * 4 + j + 2],
                            __hmul2(exp_max_old_high, out_h2[i * 4 + j + 2])),
                    thread_sum_new_high);
      }
    }
    thread_sum_old = thread_sum_new;
    thread_max_old = thread_max_new;
  }

#pragma unroll
  for (int i = 0; i < NUM_HEAD_WARPS; i++) {
#pragma unroll
    for (int j = 0; j < 4; j++) {
      out_h2[i * 4 + j] = __hmul2(out_h2[i * 4 + j], scale_o_re);
      out_i82[i * 4 + j].x = T2int8(out_h2[i * 4 + j].x);
      out_i82[i * 4 + j].y = T2int8(out_h2[i * 4 + j].y);
    }

    output_ptr[mem_lane_id_y * HEAD_DIM / 2 + mem_lane_id_x] =
        out_i82[i * 4 + 0];
    output_ptr[mem_lane_id_y * HEAD_DIM / 2 + mem_lane_id_x + 4] =
        out_i82[i * 4 + 1];
    output_ptr[mem_lane_id_y * HEAD_DIM / 2 + mem_lane_id_x +
               TC_SIZE * HEAD_DIM / 4] = out_i82[i * 4 + 2];
    output_ptr[mem_lane_id_y * HEAD_DIM / 2 + mem_lane_id_x +
               TC_SIZE * HEAD_DIM / 4 + 4] = out_i82[i * 4 + 3];

    output_ptr += TC_SIZE / 2;
  }
}
#else
template <int HEAD_DIM, int BASE_SEQ_LEN>
__global__ void
FMHAInferInt8Kernel(const int8_4 *__restrict__ query, const float scale_q,
                    const int8_4 *__restrict__ key, const float scale_k,
                    const int8_4 *__restrict__ value, const float scale_v,
                    const float sqrt_d, int8_4 *output, const float scale_o,
                    const int KV_LEN) {
  static_assert(BASE_SEQ_LEN % TC_SIZE == 0 && HEAD_DIM % TC_SIZE == 0 &&
                    BASE_SEQ_LEN >= HEAD_DIM,
                "");
  const int NUM_WARPS = BASE_SEQ_LEN / TC_SIZE;
  const int NUM_HEAD_WARPS = HEAD_DIM / TC_SIZE;

  __shared__ int share_buffer[BASE_SEQ_LEN * TC_SIZE * 5 / 4];
  auto qk_buffer_h =
      reinterpret_cast<__half *>(share_buffer + BASE_SEQ_LEN * TC_SIZE / 2);
  auto qk_buffer_i84 =
      reinterpret_cast<int8_4 *>(share_buffer + BASE_SEQ_LEN * TC_SIZE);
  auto out_buffer_h = reinterpret_cast<__half *>(share_buffer);

  const unsigned int batch = blockIdx.y;
  const unsigned int Q_LEN = gridDim.x * BASE_SEQ_LEN;
  const unsigned int q_start = blockIdx.x * BASE_SEQ_LEN;
  const unsigned int tid = threadIdx.x;
  const unsigned int warp_id = tid / WARP_SIZE;
  const unsigned int lane_id = tid % WARP_SIZE;
  const unsigned int warp_num = blockDim.x / WARP_SIZE;

  const unsigned int mem_lane_id_x = lane_id % 4;
  const unsigned int mem_lane_id_y = lane_id / 4;

  const float scale_qkv = scale_v * __frcp_rn(127.f);
  const __half2 scale_softmax_re = __float2half2_rn(127.f);
  const __half2 scale_o_re = __float2half2_rn(__frcp_rn(scale_o));

  int8_4 *smem_q[2];
  int8_4 *smem_k[2];
  int8_4 *smem_v[2];

  smem_q[0] = reinterpret_cast<int8_4 *>(share_buffer);
  smem_q[1] =
      reinterpret_cast<int8_4 *>(share_buffer + 1 * BASE_SEQ_LEN * TC_SIZE / 4);
  smem_k[0] =
      reinterpret_cast<int8_4 *>(share_buffer + 2 * BASE_SEQ_LEN * TC_SIZE / 4);
  smem_k[1] =
      reinterpret_cast<int8_4 *>(share_buffer + 3 * BASE_SEQ_LEN * TC_SIZE / 4);
  smem_v[0] =
      reinterpret_cast<int8_4 *>(share_buffer + 2 * BASE_SEQ_LEN * TC_SIZE / 4);
  smem_v[1] =
      reinterpret_cast<int8_4 *>(share_buffer + 3 * BASE_SEQ_LEN * TC_SIZE / 4);

  const int8_4 *query_ptr =
      query + batch * Q_LEN * HEAD_DIM / 4 + q_start * HEAD_DIM / 4;
  int8_4 *output_ptr =
      output + batch * Q_LEN * HEAD_DIM / 4 + q_start * HEAD_DIM / 4;

  __half2 thread_max_old =
      __half2half2(-cuda::std::numeric_limits<__half>::infinity());
  __half2 thread_sum_old = __float2half2_rn(0.f);

  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, TC_SIZE, TC_SIZE, TC_SIZE,
                         int8_t, nvcuda::wmma::row_major>
      q_frag;
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, TC_SIZE, TC_SIZE, TC_SIZE,
                         int8_t, nvcuda::wmma::col_major>
      k_frag;
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, TC_SIZE, TC_SIZE, TC_SIZE,
                         int8_t, nvcuda::wmma::row_major>
      qk_frag[NUM_WARPS];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, TC_SIZE, TC_SIZE, TC_SIZE,
                         int8_t, nvcuda::wmma::row_major>
      v_frag;
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, TC_SIZE, TC_SIZE, TC_SIZE,
                         int32_t>
      qk_out_frag_int32[NUM_WARPS > NUM_HEAD_WARPS ? NUM_WARPS
                                                   : NUM_HEAD_WARPS];
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half>
      qk_out_frag_half[NUM_WARPS];
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, TC_SIZE, TC_SIZE, TC_SIZE,
                         __half>
      out_frag[NUM_HEAD_WARPS];

#pragma unroll
  for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
    nvcuda::wmma::fill_fragment(out_frag[xi], 0);
  }

#pragma unroll
  for (int kv_start = 0; kv_start < KV_LEN; kv_start += BASE_SEQ_LEN) {
    const int8_4 *key_ptr =
        key + batch * KV_LEN * HEAD_DIM / 4 + kv_start * HEAD_DIM / 4;

    __half2 thread_max =
        __half2half2(-cuda::std::numeric_limits<__half>::infinity());
    __half2 thread_sum = __float2half2_rn(0.f);

#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
      nvcuda::wmma::fill_fragment(qk_out_frag_int32[xi], 0);
    }

    int k = 0, stride_warp = TC_SIZE * TC_SIZE / 4 / WARP_SIZE;
    int8_4 temp_int8_4;
#pragma unroll
    for (int i = 0; i < stride_warp; i++) {
      temp_int8_4 =
          *(query_ptr + warp_id * TC_SIZE * HEAD_DIM / 4 +
            i * WARP_SIZE * HEAD_DIM / TC_SIZE + mem_lane_id_y * HEAD_DIM / 4 +
            mem_lane_id_x + k * TC_SIZE / 4);
      *(smem_q[k % 2] + warp_id * TC_SIZE * TC_SIZE / 4 + i * WARP_SIZE +
        mem_lane_id_y * TC_SIZE / 4 + mem_lane_id_x) = temp_int8_4;

      temp_int8_4 =
          *(key_ptr + warp_id * TC_SIZE * HEAD_DIM / 4 +
            i * WARP_SIZE * HEAD_DIM / TC_SIZE + mem_lane_id_y * HEAD_DIM / 4 +
            mem_lane_id_x + k * TC_SIZE / 4);
      *(smem_k[k % 2] + warp_id * TC_SIZE * TC_SIZE / 4 + i * WARP_SIZE +
        mem_lane_id_y * TC_SIZE / 4 + mem_lane_id_x) = temp_int8_4;
    }

#pragma unroll
    for (k = 1; k < NUM_HEAD_WARPS; k++) {
      __syncthreads();
#pragma unroll
      for (int i = 0; i < stride_warp; i++) {
        temp_int8_4 =
            *(query_ptr + warp_id * TC_SIZE * HEAD_DIM / 4 +
              i * WARP_SIZE * HEAD_DIM / TC_SIZE +
              mem_lane_id_y * HEAD_DIM / 4 + mem_lane_id_x + k * TC_SIZE / 4);
        *(smem_q[k % 2] + warp_id * TC_SIZE * TC_SIZE / 4 + i * WARP_SIZE +
          mem_lane_id_y * TC_SIZE / 4 + mem_lane_id_x) = temp_int8_4;

        temp_int8_4 =
            *(key_ptr + warp_id * TC_SIZE * HEAD_DIM / 4 +
              i * WARP_SIZE * HEAD_DIM / TC_SIZE +
              mem_lane_id_y * HEAD_DIM / 4 + mem_lane_id_x + k * TC_SIZE / 4);
        *(smem_k[k % 2] + warp_id * TC_SIZE * TC_SIZE / 4 + i * WARP_SIZE +
          mem_lane_id_y * TC_SIZE / 4 + mem_lane_id_x) = temp_int8_4;
      }

      nvcuda::wmma::load_matrix_sync(
          q_frag,
          reinterpret_cast<int8_t *>(
              &(smem_q[(k - 1) % 2][warp_id * TC_SIZE * TC_SIZE / 4])),
          TC_SIZE);
#pragma unroll
      for (int xi = 0; xi < NUM_WARPS; xi++) {
        nvcuda::wmma::load_matrix_sync(
            k_frag,
            reinterpret_cast<int8_t *>(
                &(smem_k[(k - 1) % 2][xi * TC_SIZE * TC_SIZE / 4])),
            TC_SIZE);
        nvcuda::wmma::mma_sync(qk_out_frag_int32[xi], q_frag, k_frag,
                               qk_out_frag_int32[xi]);
      }
    }
    __syncthreads();
    k = NUM_HEAD_WARPS - 1;
    nvcuda::wmma::load_matrix_sync(
        q_frag,
        reinterpret_cast<int8_t *>(
            &(smem_q[k % 2][warp_id * TC_SIZE * TC_SIZE / 4])),
        TC_SIZE);
#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
      nvcuda::wmma::load_matrix_sync(
          k_frag,
          reinterpret_cast<int8_t *>(
              &(smem_k[k % 2][xi * TC_SIZE * TC_SIZE / 4])),
          TC_SIZE);
      nvcuda::wmma::mma_sync(qk_out_frag_int32[xi], q_frag, k_frag,
                             qk_out_frag_int32[xi]);
    }

#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
#pragma unroll
      for (int tc_xi = 0; tc_xi < 2; tc_xi++) {
        __half2 temp1 =
            __floats2half2_rn(qk_out_frag_int32[xi].x[tc_xi * 4] * sqrt_d,
                              qk_out_frag_int32[xi].x[tc_xi * 4 + 2] * sqrt_d);
        __half2 temp2 =
            __floats2half2_rn(qk_out_frag_int32[xi].x[tc_xi * 4 + 1] * sqrt_d,
                              qk_out_frag_int32[xi].x[tc_xi * 4 + 3] * sqrt_d);

        qk_out_frag_half[xi].x[tc_xi * 4] = __low2half(temp1);
        qk_out_frag_half[xi].x[tc_xi * 4 + 2] = __high2half(temp1);
        qk_out_frag_half[xi].x[tc_xi * 4 + 1] = __low2half(temp2);
        qk_out_frag_half[xi].x[tc_xi * 4 + 3] = __high2half(temp2);
        thread_max = hmax(thread_max, hmax(temp1, temp2));
      }
    }

#pragma unroll
    for (int s = 2; s > 0; s >>= 1) {
      thread_max =
          hmax(thread_max, __shfl_xor_sync(0xffffffff, thread_max, s, 4));
    }

#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
#pragma unroll
      for (int tc_xi = 0; tc_xi < 2; tc_xi++) {
        __half2 temp1 = __halves2half2(qk_out_frag_half[xi].x[tc_xi * 4],
                                       qk_out_frag_half[xi].x[tc_xi * 4 + 2]);
        __half2 temp2 = __halves2half2(qk_out_frag_half[xi].x[tc_xi * 4 + 1],
                                       qk_out_frag_half[xi].x[tc_xi * 4 + 3]);
        temp1 = h2exp(__hsub2(temp1, thread_max));
        temp2 = h2exp(__hsub2(temp2, thread_max));
        qk_out_frag_half[xi].x[tc_xi * 4] = __low2half(temp1);
        qk_out_frag_half[xi].x[tc_xi * 4 + 2] = __high2half(temp1);
        qk_out_frag_half[xi].x[tc_xi * 4 + 1] = __low2half(temp2);
        qk_out_frag_half[xi].x[tc_xi * 4 + 3] = __high2half(temp2);
        thread_sum = __hadd2(thread_sum, __hadd2(temp1, temp2));
      }
    }

#pragma unroll
    for (int s = 2; s > 0; s >>= 1) {
      thread_sum =
          __hadd2(thread_sum, __shfl_xor_sync(0xffffffff, thread_sum, s, 4));
    }

    __syncthreads();
#pragma unroll
    for (int xi = 0; xi < NUM_WARPS; xi++) {
      nvcuda::wmma::store_matrix_sync(qk_buffer_h + warp_id * TC_SIZE * TC_SIZE,
                                      qk_out_frag_half[xi], TC_SIZE,
                                      nvcuda::wmma::mem_row_major);
      for (int i = 0; i < TC_SIZE * TC_SIZE / WARP_SIZE / 4; i++) {
        __half2 t1 = *reinterpret_cast<__half2 *>(
            qk_buffer_h + warp_id * TC_SIZE * TC_SIZE + i * WARP_SIZE * 4 +
            lane_id * 4);
        __half2 t2 = *reinterpret_cast<__half2 *>(
            qk_buffer_h + warp_id * TC_SIZE * TC_SIZE + i * WARP_SIZE * 4 +
            lane_id * 4 + 2);

        t1 = __hmul2(t1, scale_softmax_re);
        t2 = __hmul2(t2, scale_softmax_re);
        int8_4 t3 = int8_4(T2int8(__low2half(t1)), T2int8(__high2half(t1)),
                           T2int8(__low2half(t2)), T2int8(__high2half(t2)));
        qk_buffer_i84[warp_id * TC_SIZE * TC_SIZE / 4 + i * WARP_SIZE +
                      lane_id] = t3;
      }
      nvcuda::wmma::load_matrix_sync(
          qk_frag[xi],
          reinterpret_cast<int8_t *>(qk_buffer_i84 +
                                     warp_id * TC_SIZE * TC_SIZE / 4),
          TC_SIZE);
    }
    __syncthreads();

#pragma unroll
    for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
      nvcuda::wmma::fill_fragment(qk_out_frag_int32[xi], 0);
    }

    stride_warp = TC_SIZE / warp_num;
    const int8_4 *value_ptr = value + batch * KV_LEN * HEAD_DIM / 4 +
                              (kv_start + stride_warp * warp_id) * HEAD_DIM / 4;
    k = 0;
#pragma unroll
    for (int i = 0; i < stride_warp * HEAD_DIM / 4 / WARP_SIZE; i++) {
      temp_int8_4 =
          *(value_ptr + k * TC_SIZE * HEAD_DIM / 4 + i * WARP_SIZE + lane_id);
      *(smem_v[k % 2] + warp_id * stride_warp * HEAD_DIM / 4 + i * WARP_SIZE +
        lane_id) = temp_int8_4;
    }
#pragma unroll
    for (k = 1; k < NUM_WARPS; k++) {
      __syncthreads();
      for (int i = 0; i < stride_warp * HEAD_DIM / 4 / WARP_SIZE; i++) {
        temp_int8_4 =
            *(value_ptr + k * TC_SIZE * HEAD_DIM / 4 + i * WARP_SIZE + lane_id);
        *(smem_v[k % 2] + warp_id * stride_warp * HEAD_DIM / 4 + i * WARP_SIZE +
          lane_id) = temp_int8_4;
      }
#pragma unroll
      for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
        nvcuda::wmma::load_matrix_sync(
            v_frag,
            reinterpret_cast<int8_t *>(
                &(smem_v[(k - 1) % 2][xi * TC_SIZE / 4])),
            HEAD_DIM);
        nvcuda::wmma::mma_sync(qk_out_frag_int32[xi], qk_frag[k - 1], v_frag,
                               qk_out_frag_int32[xi]);
      }
    }
    __syncthreads();
    k = NUM_WARPS - 1;
#pragma unroll
    for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
      nvcuda::wmma::load_matrix_sync(
          v_frag,
          reinterpret_cast<int8_t *>(&(smem_v[k % 2][xi * TC_SIZE / 4])),
          HEAD_DIM);
      nvcuda::wmma::mma_sync(qk_out_frag_int32[xi], qk_frag[k], v_frag,
                             qk_out_frag_int32[xi]);
    }

    __half2 thread_max_new = hmax(thread_max_old, thread_max);
    __half2 exp_max_old = h2exp(__hsub2(thread_max_old, thread_max_new));
    __half2 exp_max = h2exp(__hsub2(thread_max, thread_max_new));
    __half2 thread_sum_new = __hadd2(__hmul2(exp_max_old, thread_sum_old),
                                     __hmul2(exp_max, thread_sum));

#pragma unroll
    for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
#pragma unroll
      for (int tc_xi = 0; tc_xi < 2; tc_xi++) {
        __half2 temp1 = __halves2half2(out_frag[xi].x[tc_xi * 4],
                                       out_frag[xi].x[tc_xi * 4 + 2]);
        __half2 temp2 = __halves2half2(out_frag[xi].x[tc_xi * 4 + 1],
                                       out_frag[xi].x[tc_xi * 4 + 3]);
        __half2 temp1_kv = __floats2half2_rn(
            qk_out_frag_int32[xi].x[tc_xi * 4] * scale_qkv,
            qk_out_frag_int32[xi].x[tc_xi * 4 + 2] * scale_qkv);
        __half2 temp2_kv = __floats2half2_rn(
            qk_out_frag_int32[xi].x[tc_xi * 4 + 1] * scale_qkv,
            qk_out_frag_int32[xi].x[tc_xi * 4 + 3] * scale_qkv);

        temp1 = __h2div(
            __hfma2(exp_max, temp1_kv,
                    __hmul2(__hmul2(thread_sum_old, exp_max_old), temp1)),
            thread_sum_new);
        temp2 = __h2div(
            __hfma2(exp_max, temp2_kv,
                    __hmul2(__hmul2(thread_sum_old, exp_max_old), temp2)),
            thread_sum_new);

        out_frag[xi].x[tc_xi * 4] = __low2half(temp1);
        out_frag[xi].x[tc_xi * 4 + 2] = __high2half(temp1);
        out_frag[xi].x[tc_xi * 4 + 1] = __low2half(temp2);
        out_frag[xi].x[tc_xi * 4 + 3] = __high2half(temp2);
      }
    }
    thread_sum_old = thread_sum_new;
    thread_max_old = thread_max_new;
  }

#pragma unroll
  for (int xi = 0; xi < NUM_HEAD_WARPS; xi++) {
    nvcuda::wmma::store_matrix_sync(out_buffer_h + warp_id * TC_SIZE * TC_SIZE,
                                    out_frag[xi], TC_SIZE,
                                    nvcuda::wmma::mem_row_major);
#pragma unroll
    for (int i = 0; i < TC_SIZE * TC_SIZE / WARP_SIZE / 4; i++) {
      __half2 t1 = *reinterpret_cast<__half2 *>(
          out_buffer_h + warp_id * TC_SIZE * TC_SIZE + i * WARP_SIZE * 4 +
          lane_id * 4);
      __half2 t2 = *reinterpret_cast<__half2 *>(
          out_buffer_h + warp_id * TC_SIZE * TC_SIZE + i * WARP_SIZE * 4 +
          lane_id * 4 + 2);

      t1 = __hmul2(t1, scale_o_re);
      t2 = __hmul2(t2, scale_o_re);

      int8_4 t3 = int8_4(T2int8(__low2half(t1)), T2int8(__high2half(t1)),
                         T2int8(__low2half(t2)), T2int8(__high2half(t2)));
      *(output_ptr + warp_id * TC_SIZE * HEAD_DIM / 4 + xi * TC_SIZE / 4 +
        i * HEAD_DIM * WARP_SIZE / TC_SIZE + mem_lane_id_y * HEAD_DIM / 4 +
        mem_lane_id_x) = t3;
    }
  }
}
#endif

template <typename T>
int qkv_flash(const T *query, const T *key, const T *value, T *output,
              const int &batch, const int &q_len, const int &kv_len,
              const int &embed_dim, hipStream_t stream) {
  const T sqrt_d = 1.f / std::sqrt((float)embed_dim);

#define HEAD_DEFINE(BASE_LEN)                                                  \
  if (embed_dim == 32) {                                                       \
    FMHAInferKernel<32, BASE_LEN>                                              \
        <<<dim3(q_len / BASE_LEN, batch), BASE_LEN * 2, 0, stream>>>(          \
            query, key, value, sqrt_d, output, kv_len);                        \
  } else if (embed_dim == 64) {                                                \
    FMHAInferKernel<64, BASE_LEN>                                              \
        <<<dim3(q_len / BASE_LEN, batch), BASE_LEN * 2, 0, stream>>>(          \
            query, key, value, sqrt_d, output, kv_len);                        \
  } else {                                                                     \
    printf("Do not support head_dim=%d\n", embed_dim);                         \
    exit(1);                                                                   \
  }

  if (q_len % 128 == 0 && kv_len % 128 == 0) {
    HEAD_DEFINE(128)
  } else if (q_len % 64 == 0 && kv_len % 64 == 0) {
    HEAD_DEFINE(64)
  } else {
    printf("Do not support q_len=%d, kv_len=%d\n", q_len, kv_len);
    exit(1);
  }
  cudaCheckError();
  return 0;
}

template <>
int qkv_flash(const __half2 *query, const __half2 *key, const __half2 *value,
              __half2 *output, const int &batch, const int &q_len,
              const int &kv_len, const int &embed_dim, hipStream_t stream) {
  const __half2 sqrt_d = __float2half2_rn(1.f / std::sqrt((float)embed_dim));

#define HEAD_DEFINE(BASE_LEN)                                                  \
  if (embed_dim == 32) {                                                       \
    FMHAInferKernel<32, BASE_LEN>                                              \
        <<<dim3(q_len / BASE_LEN, batch), BASE_LEN * 2, 0, stream>>>(          \
            query, key, value, sqrt_d, output, kv_len);                        \
  } else if (embed_dim == 64) {                                                \
    FMHAInferKernel<64, BASE_LEN>                                              \
        <<<dim3(q_len / BASE_LEN, batch), BASE_LEN * 2, 0, stream>>>(          \
            query, key, value, sqrt_d, output, kv_len);                        \
  } else {                                                                     \
    printf("Do not support head_dim=%d\n", embed_dim);                         \
    exit(1);                                                                   \
  }

  if (q_len % 128 == 0 && kv_len % 128 == 0) {
    HEAD_DEFINE(128)
  } else if (q_len % 64 == 0 && kv_len % 64 == 0) {
    HEAD_DEFINE(64)
  } else {
    printf("Do not support q_len=%d, kv_len=%d\n", q_len, kv_len);
    exit(1);
  }
  cudaCheckError();
  return 0;
}

int qkv_flash_int8(const int8_4 *query, const float &scale_q, const int8_4 *key,
                   const float &scale_k, const int8_4 *value,
                   const float &scale_v, int8_4 *output, const float &scale_o,
                   const int &batch, const int &q_len, const int &kv_len,
                   const int &embed_dim, hipStream_t stream) {
  const float sqrt_d = scale_q * scale_k / std::sqrt((float)embed_dim);

#define HEAD_DEFINE_INT8(BASE_LEN)                                             \
  if (embed_dim == 32) {                                                       \
    FMHAInferInt8Kernel<32, BASE_LEN>                                          \
        <<<dim3(q_len / BASE_LEN, batch), BASE_LEN * 2, 0, stream>>>(          \
            query, scale_q, key, scale_k, value, scale_v, sqrt_d, output,      \
            scale_o, kv_len);                                                  \
  } else if (embed_dim == 64) {                                                \
    FMHAInferInt8Kernel<64, BASE_LEN>                                          \
        <<<dim3(q_len / BASE_LEN, batch), BASE_LEN * 2, 0, stream>>>(          \
            query, scale_q, key, scale_k, value, scale_v, sqrt_d, output,      \
            scale_o, kv_len);                                                  \
  } else {                                                                     \
    printf("Do not support head_dim=%d\n", embed_dim);                         \
    exit(1);                                                                   \
  }

  if (q_len % 128 == 0 && kv_len % 128 == 0) {
    HEAD_DEFINE_INT8(128)
  } else if (q_len % 64 == 0 && kv_len % 64 == 0) {
    HEAD_DEFINE_INT8(64)
  } else {
    printf("Do not support q_len=%d, kv_len=%d\n", q_len, kv_len);
    exit(1);
  }
  cudaCheckError();
  return 0;
}

template int qkv_flash(const float *query, const float *key, const float *value,
                       float *output, const int &batch, const int &q_len,
                       const int &kv_len, const int &embed_dim,
                       hipStream_t stream);
template int qkv_flash(const __half *query, const __half *key,
                       const __half *value, __half *output, const int &batch,
                       const int &q_len, const int &kv_len,
                       const int &embed_dim, hipStream_t stream);
template int qkv_flash(const __half2 *query, const __half2 *key,
                       const __half2 *value, __half2 *output, const int &batch,
                       const int &q_len, const int &kv_len,
                       const int &embed_dim, hipStream_t stream);
