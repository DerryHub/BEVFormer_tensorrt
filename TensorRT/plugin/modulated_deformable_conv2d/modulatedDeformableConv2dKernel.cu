#include "hip/hip_runtime.h"
//
// Created by Derry Lin on 2022/11/10.
//

#include <cstdio>
#include <hip/hip_fp16.h>

#include <algorithm>
#include <cmath>
#include <vector>

#include "cuda_helper.h"
#include "helper.h"
#include "modulatedDeformableConv2dKernel.h"
#include <cstdio>
#include <unistd.h>

template <typename T> __forceinline__ __device__ T sign_05(T x) {
  if (x > 0) {
    return 0.5f;
  }
  return -0.5f;
}

template <typename T> __forceinline__ __device__ int8_t T2int8(T a) {
  a = a > 127 ? 127 : a;
  a = a < -128 ? -128 : a;
  return int8_t(a + sign_05<T>(a));
}

template <> __forceinline__ __device__ int8_t T2int8(__half a) {
  a = __hgt(a, __int2half_rn(127)) ? __int2half_rn(127) : a;
  a = __hlt(a, __int2half_rn(-128)) ? __int2half_rn(-128) : a;
  return int8_t(__half2int_rn(a));
}

__forceinline__ __device__ int8_t half2int8(const __half &hval,
                                            const float &scale) {
  __half ret = __hdiv(hval, __float2half(scale));
  return T2int8<__half>(ret);
}

__forceinline__ __device__ void qmulf(const int8_4 &a, int8_4 &c,
                                      const float &b) {
  c.x = T2int8<float>(a.x * b);
  c.y = T2int8<float>(a.y * b);
  c.z = T2int8<float>(a.z * b);
  c.w = T2int8<float>(a.w * b);
}

__forceinline__ __device__ void dp4a(const int32_t *a, const int32_t *b,
                                     int32_t &c) {
#if __CUDA_ARCH__ >= 610
  asm("dp4a.s32.s32 %0, %1, %2, %3;" : "+r"(c) : "r"(*a), "r"(*b), "r"(c));
#else
  auto ap = (int8_4 *)a, bp = (int8_4 *)b;

  c += ap->x * bp->x;
  c += ap->y * bp->y;
  c += ap->z * bp->z;
  c += ap->w * bp->w;
#endif
}

template <typename scalar_t>
__device__ scalar_t dmcn_im2col_bilinear(const scalar_t *input,
                                         const int data_width, const int height,
                                         const int width, scalar_t h,
                                         scalar_t w) {
  int h_low = floorf(h);
  int w_low = floorf(w);
  int h_high = h_low + 1;
  int w_high = w_low + 1;

  scalar_t lh = h - h_low;
  scalar_t lw = w - w_low;
  scalar_t hh = 1 - lh, hw = 1 - lw;

  scalar_t v1 = 0;
  if (h_low >= 0 && w_low >= 0)
    v1 = input[h_low * data_width + w_low];
  scalar_t v2 = 0;
  if (h_low >= 0 && w_high <= width - 1)
    v2 = input[h_low * data_width + w_high];
  scalar_t v3 = 0;
  if (h_high <= height - 1 && w_low >= 0)
    v3 = input[h_high * data_width + w_low];
  scalar_t v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
    v4 = input[h_high * data_width + w_high];

  scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <>
__device__ __half dmcn_im2col_bilinear(const __half *input,
                                       const int data_width, const int height,
                                       const int width, __half h, __half w) {
  __half h_low = hfloor(h);
  __half w_low = hfloor(w);
  __half h_high = __hadd(h_low, __float2half(1.f));
  __half w_high = __hadd(w_low, __float2half(1.f));

  __half lh = __hsub(h, h_low);
  __half lw = __hsub(w, w_low);
  __half hh = __hsub(__float2half(1.f), lh), hw = __hsub(__float2half(1.f), lw);

  __half v1 = __float2half(0.f);
  if (__hge(h_low, __float2half(0.f)) && __hge(w_low, __float2half(0.f)))
    v1 = input[static_cast<int>(h_low) * data_width + static_cast<int>(w_low)];
  __half v2 = __float2half(0.f);
  if (__hge(h_low, __float2half(0.f)) &&
      __hle(w_high, __float2half(static_cast<float>(width - 1))))
    v2 = input[static_cast<int>(h_low) * data_width + static_cast<int>(w_high)];
  __half v3 = __float2half(0.f);
  if (__hle(h_high, __float2half(static_cast<float>(height - 1))) &&
      __hge(w_low, __float2half(0.f)))
    v3 = input[static_cast<int>(h_high) * data_width + static_cast<int>(w_low)];
  __half v4 = __float2half(0.f);
  if (__hle(h_high, __float2half(static_cast<float>(height - 1))) &&
      __hle(w_high, __float2half(static_cast<float>(width - 1))))
    v4 =
        input[static_cast<int>(h_high) * data_width + static_cast<int>(w_high)];

  __half w1 = __hmul(hh, hw), w2 = __hmul(hh, lw), w3 = __hmul(lh, hw),
         w4 = __hmul(lh, lw);

  __half val = __hadd(__hadd(__hmul(w1, v1), __hmul(w2, v2)),
                      __hadd(__hmul(w3, v3), __hmul(w4, v4)));
  return val;
}

__device__ __half2 dmcn_im2col_bilinear_h2(const __half2 *input,
                                           const int data_width,
                                           const int height, const int width,
                                           __half2 hw) {
  __half2 hw_low = h2floor(hw);
  __half2 hw_high = __hadd2(hw_low, __float2half2_rn(1.f));
  __half2 lhw = __hsub2(hw, hw_low);
  __half2 hhw = __hsub2(__float2half2_rn(1.f), lhw);
  __half2 w1 = __half2half2(__hmul(__low2half(hhw), __high2half(hhw))),
          w2 = __half2half2(__hmul(__low2half(hhw), __high2half(lhw))),
          w3 = __half2half2(__hmul(__low2half(lhw), __high2half(hhw))),
          w4 = __half2half2(__hmul(__low2half(lhw), __high2half(lhw)));

  int h_low = static_cast<int>(__low2float(hw_low)),
      w_low = static_cast<int>(__high2float(hw_low)),
      h_high = static_cast<int>(__low2float(hw_high)),
      w_high = static_cast<int>(__high2float(hw_high));
  __half2 val = __float2half2_rn(0.f);

  if (h_low >= 0 && w_low >= 0) {
    val = __hfma2(input[h_low * data_width + w_low], w1, val);
  }
  if (h_low >= 0 && w_high < width) {
    val = __hfma2(input[h_low * data_width + w_high], w2, val);
  }
  if (h_high < height && w_low >= 0) {
    val = __hfma2(input[h_high * data_width + w_low], w3, val);
  }
  if (h_high < height && w_high < width) {
    val = __hfma2(input[h_high * data_width + w_high], w4, val);
  }
  return val;
}

__forceinline__ __device__ void
dmcn_im2col_bilinear_int8(const int8_4 *input, const float &scale_i,
                          const int data_width, const int height,
                          const int width, __half2 hw, int8_4 &output) {
  __half2 hw_low = h2floor(hw);
  __half2 hw_high = __hadd2(hw_low, __float2half2_rn(1.f));
  __half2 lhw = __hsub2(hw, hw_low);
  __half2 hhw = __hsub2(__float2half2_rn(1.f), lhw);

  const float scale_area = 1 / 127.f;
  int8_4 weight = {
      half2int8(__hmul(__low2half(hhw), __high2half(hhw)), scale_area),
      half2int8(__hmul(__low2half(hhw), __high2half(lhw)), scale_area),
      half2int8(__hmul(__low2half(lhw), __high2half(hhw)), scale_area),
      half2int8(__hmul(__low2half(lhw), __high2half(lhw)), scale_area)};

  int h_low = static_cast<int>(__low2float(hw_low)),
      w_low = static_cast<int>(__high2float(hw_low)),
      h_high = static_cast<int>(__low2float(hw_high)),
      w_high = static_cast<int>(__high2float(hw_high));
  int8_4 inps[4] = {0, 0, 0, 0};
  int32_t output_temp;

  if (h_low >= 0 && w_low >= 0) {
    const int8_4 &inp = input[h_low * data_width + w_low];
    inps[0].x = inp.x;
    inps[1].x = inp.y;
    inps[2].x = inp.z;
    inps[3].x = inp.w;
  }
  if (h_low >= 0 && w_high < width) {
    const int8_4 &inp = input[h_low * data_width + w_high];
    inps[0].y = inp.x;
    inps[1].y = inp.y;
    inps[2].y = inp.z;
    inps[3].y = inp.w;
  }
  if (h_high < height && w_low >= 0) {
    const int8_4 &inp = input[h_high * data_width + w_low];
    inps[0].z = inp.x;
    inps[1].z = inp.y;
    inps[2].z = inp.z;
    inps[3].z = inp.w;
  }
  if (h_high < height && w_high < width) {
    const int8_4 &inp = input[h_high * data_width + w_high];
    inps[0].w = inp.x;
    inps[1].w = inp.y;
    inps[2].w = inp.z;
    inps[3].w = inp.w;
  }

  output_temp = 0;
  dp4a((const int32_t *)inps, (const int32_t *)&weight, output_temp);
  output.x = T2int8<float>(output_temp * scale_area);
  ;

  output_temp = 0;
  dp4a((const int32_t *)(inps + 1), (const int32_t *)&weight, output_temp);
  output.y = T2int8<float>(output_temp * scale_area);

  output_temp = 0;
  dp4a((const int32_t *)(inps + 2), (const int32_t *)&weight, output_temp);
  output.z = T2int8<float>(output_temp * scale_area);

  output_temp = 0;
  dp4a((const int32_t *)(inps + 3), (const int32_t *)&weight, output_temp);
  output.w = T2int8<float>(output_temp * scale_area);
}

template <typename scalar_t>
__global__ void modulated_deformable_im2col_gpu_kernel(
    const int n, const scalar_t *data_im, const scalar_t *data_offset,
    const scalar_t *data_mask, const int height, const int width,
    const int kernel_h, const int kernel_w, const int pad_h, const int pad_w,
    const int stride_h, const int stride_w, const int dilation_h,
    const int dilation_w, const int channel_per_deformable_group,
    const int batch_size, const int num_channels, const int deformable_group,
    const int height_col, const int width_col, scalar_t *data_col) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    // index of output matrix
    const int w_col = index % width_col;
    const int h_col = (index / width_col) % height_col;
    const int b_col = (index / width_col / height_col) % batch_size;
    const int c_im = (index / width_col / height_col) / batch_size;
    const int c_col = c_im * kernel_h * kernel_w;

    // compute deformable group index
    const int deformable_group_index = c_im / channel_per_deformable_group;

    const int h_in = h_col * stride_h - pad_h;
    const int w_in = w_col * stride_w - pad_w;

    scalar_t *data_col_ptr =
        data_col +
        ((c_col * batch_size + b_col) * height_col + h_col) * width_col + w_col;
    const scalar_t *data_im_ptr =
        data_im + (b_col * num_channels + c_im) * height * width;
    const scalar_t *data_offset_ptr =
        data_offset + (b_col * deformable_group + deformable_group_index) * 2 *
                          kernel_h * kernel_w * height_col * width_col;

    const scalar_t *data_mask_ptr =
        data_mask + (b_col * deformable_group + deformable_group_index) *
                        kernel_h * kernel_w * height_col * width_col;

    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        const int data_offset_h_ptr =
            ((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col;
        const int data_offset_w_ptr =
            ((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col +
            w_col;
        const int data_mask_hw_ptr =
            ((i * kernel_w + j) * height_col + h_col) * width_col + w_col;
        const scalar_t offset_h = data_offset_ptr[data_offset_h_ptr];
        const scalar_t offset_w = data_offset_ptr[data_offset_w_ptr];
        const scalar_t mask = data_mask_ptr[data_mask_hw_ptr];
        auto val = static_cast<scalar_t>(0);
        const scalar_t h_im = h_in + i * dilation_h + offset_h;
        const scalar_t w_im = w_in + j * dilation_w + offset_w;
        if (h_im > -1 && w_im > -1 && h_im < height && w_im < width)
          val = dmcn_im2col_bilinear(data_im_ptr, width, height, width, h_im,
                                     w_im);
        *data_col_ptr = val * mask;
        data_col_ptr += batch_size * height_col * width_col;
      }
    }
  }
}

template <>
__global__ void modulated_deformable_im2col_gpu_kernel(
    const int n, const __half *data_im, const __half *data_offset,
    const __half *data_mask, const int height, const int width,
    const int kernel_h, const int kernel_w, const int pad_h, const int pad_w,
    const int stride_h, const int stride_w, const int dilation_h,
    const int dilation_w, const int channel_per_deformable_group,
    const int batch_size, const int num_channels, const int deformable_group,
    const int height_col, const int width_col, __half *data_col) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    // index of output matrix
    const int w_col = index % width_col;
    const int h_col = (index / width_col) % height_col;
    const int b_col = (index / width_col / height_col) % batch_size;
    const int c_im = (index / width_col / height_col) / batch_size;
    const int c_col = c_im * kernel_h * kernel_w;

    // compute deformable group index
    const int deformable_group_index = c_im / channel_per_deformable_group;

    const int h_in = h_col * stride_h - pad_h;
    const int w_in = w_col * stride_w - pad_w;

    __half *data_col_ptr =
        data_col +
        ((c_col * batch_size + b_col) * height_col + h_col) * width_col + w_col;
    const __half *data_im_ptr =
        data_im + (b_col * num_channels + c_im) * height * width;
    const __half *data_offset_ptr =
        data_offset + (b_col * deformable_group + deformable_group_index) * 2 *
                          kernel_h * kernel_w * height_col * width_col;

    const __half *data_mask_ptr =
        data_mask + (b_col * deformable_group + deformable_group_index) *
                        kernel_h * kernel_w * height_col * width_col;

    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        const int data_offset_h_ptr =
            ((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col;
        const int data_offset_w_ptr =
            ((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col +
            w_col;
        const int data_mask_hw_ptr =
            ((i * kernel_w + j) * height_col + h_col) * width_col + w_col;
        const __half offset_h = data_offset_ptr[data_offset_h_ptr];
        const __half offset_w = data_offset_ptr[data_offset_w_ptr];
        const __half mask = data_mask_ptr[data_mask_hw_ptr];
        __half val = __float2half(0.f);
        const __half h_im = __hadd(
            offset_h, __hfma(__float2half(static_cast<float>(i)),
                             __float2half(static_cast<float>(dilation_h)),
                             __float2half(static_cast<float>(h_in))));
        const __half w_im = __hadd(
            offset_w, __hfma(__float2half(static_cast<float>(j)),
                             __float2half(static_cast<float>(dilation_w)),
                             __float2half(static_cast<float>(w_in))));
        if (__hgt(h_im, __float2half(-1.f)) &&
            __hgt(w_im, __float2half(-1.f)) &&
            __hlt(h_im, __float2half(static_cast<float>(height))) &&
            __hlt(w_im, __float2half(static_cast<float>(width))))
          val = dmcn_im2col_bilinear(data_im_ptr, width, height, width, h_im,
                                     w_im);
        *data_col_ptr = __hmul(val, mask);
        data_col_ptr += batch_size * height_col * width_col;
      }
    }
  }
}

__global__ void modulated_deformable_im2col_gpu_kernel_h2(
    const int n, const __half2 *data_im, const __half2 *data_offset,
    const __half *data_mask, const int height, const int width,
    const int kernel_h, const int kernel_w, const int pad_h, const int pad_w,
    const int stride_h, const int stride_w, const int dilation_h,
    const int dilation_w, const int channel_per_deformable_group,
    const int batch_size, const int num_channels, const int deformable_group,
    const int height_col, const int width_col, __half2 *data_col) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    // index of output matrix
    const int w_col = index % width_col;
    const int h_col = (index / width_col) % height_col;
    const int b_col = (index / width_col / height_col) % batch_size;
    const int c_im = (index / width_col / height_col) / batch_size;
    const int c_col = c_im * kernel_h * kernel_w;

    // compute deformable group index
    const int deformable_group_index = c_im / channel_per_deformable_group;

    const int h_in = h_col * stride_h - pad_h;
    const int w_in = w_col * stride_w - pad_w;

    auto *data_col_ptr =
        (__half *)data_col +
        ((c_col * 2 * batch_size + b_col * 2) * height_col + h_col) *
            width_col +
        w_col;
    const __half2 *data_im_ptr =
        data_im + (b_col * ((num_channels + 1) / 2) + c_im) * height * width;
    const __half2 *data_offset_ptr =
        data_offset + (b_col * deformable_group + deformable_group_index) *
                          kernel_h * kernel_w * height_col * width_col;

    const __half *data_mask_ptr =
        data_mask + (b_col * deformable_group + deformable_group_index) *
                        kernel_h * kernel_w * height_col * width_col;

    __half2 condition;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        const int data_offset_hw_ptr =
            ((i * kernel_w + j) * height_col + h_col) * width_col + w_col;
        const int data_mask_hw_ptr =
            ((i * kernel_w + j) * height_col + h_col) * width_col + w_col;
        const __half2 offset_hw = data_offset_ptr[data_offset_hw_ptr];
        const __half2 mask = __half2half2(data_mask_ptr[data_mask_hw_ptr]);
        __half2 val = __float2half2_rn(0.f);
        const __half2 hw_im =
            __hadd2(offset_hw,
                    __hfma2(__floats2half2_rn(static_cast<float>(i),
                                              static_cast<float>(j)),
                            __floats2half2_rn(static_cast<float>(dilation_h),
                                              static_cast<float>(dilation_w)),
                            __floats2half2_rn(static_cast<float>(h_in),
                                              static_cast<float>(w_in))));
        condition = __hmul2(
            __hgt2(hw_im, __float2half2_rn(-1.f)),
            __hlt2(hw_im, __floats2half2_rn(static_cast<float>(height),
                                            static_cast<float>(width))));
        if (__low2float(condition) * __high2float(condition)) {
          val =
              dmcn_im2col_bilinear_h2(data_im_ptr, width, height, width, hw_im);
        }
        val = __hmul2(val, mask);
        *data_col_ptr = __low2half(val);
        *(data_col_ptr + batch_size * height_col * width_col) =
            __high2half(val);
        data_col_ptr += batch_size * height_col * width_col * 2;
      }
    }
  }
}

__global__ void modulated_deformable_im2col_gpu_kernel_int8(
    const int n, const int8_4 *data_im, const float scale_i,
    const int8_t *data_offset, const float scale_off, const int8_t *data_mask,
    const float scale_mask, const int height, const int width,
    const int kernel_h, const int kernel_w, const int pad_h, const int pad_w,
    const int stride_h, const int stride_w, const int dilation_h,
    const int dilation_w, const int channel_per_deformable_group,
    const int batch_size, const int num_channels, const int deformable_group,
    const int height_col, const int width_col, const int hw4,
    int8_4 *data_col) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    // index of output matrix
    const int w_col = index % width_col;
    const int h_col = (index / width_col) % height_col;
    const int b_col = (index / width_col / height_col) % batch_size;
    const int c_im = (index / width_col / height_col) / batch_size;
    const int c_col = c_im * kernel_h * kernel_w;

    // compute deformable group index
    const int deformable_group_index = c_im / channel_per_deformable_group;

    const int h_in = h_col * stride_h - pad_h;
    const int w_in = w_col * stride_w - pad_w;

    auto *data_col_ptr = (int8_t *)data_col +
                         (c_col * 4 * batch_size + b_col * 4) * hw4 +
                         h_col * width_col + w_col;
    const int8_4 *data_im_ptr =
        data_im + (b_col * ((num_channels + 3) / 4) + c_im) * height * width;
    const int8_t *data_offset_ptr =
        data_offset + (b_col * deformable_group + deformable_group_index) *
                          kernel_h * kernel_w * height_col * width_col * 2;

    const int8_t *data_mask_ptr =
        data_mask + (b_col * deformable_group + deformable_group_index) *
                        kernel_h * kernel_w * height_col * width_col;
    const int32_t output_step = batch_size * hw4;

    __half2 condition;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        const int data_offset_h_ptr =
            ((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col;
        const int data_offset_w_ptr =
            ((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col +
            w_col;
        const int data_mask_hw_ptr =
            ((i * kernel_w + j) * height_col + h_col) * width_col + w_col;

        const __half2 offset_hw =
            __floats2half2_rn(data_offset_ptr[data_offset_h_ptr] * scale_off,
                              data_offset_ptr[data_offset_w_ptr] * scale_off);

        const float mask = data_mask_ptr[data_mask_hw_ptr] * scale_mask;

        const __half2 hw_im =
            __hadd2(offset_hw,
                    __hfma2(__floats2half2_rn(static_cast<float>(i),
                                              static_cast<float>(j)),
                            __floats2half2_rn(static_cast<float>(dilation_h),
                                              static_cast<float>(dilation_w)),
                            __floats2half2_rn(static_cast<float>(h_in),
                                              static_cast<float>(w_in))));
        condition = __hmul2(
            __hgt2(hw_im, __float2half2_rn(-1.f)),
            __hlt2(hw_im, __floats2half2_rn(static_cast<float>(height),
                                            static_cast<float>(width))));
        int8_4 val = 0;
        if (__low2float(condition) * __high2float(condition)) {
          dmcn_im2col_bilinear_int8(data_im_ptr, scale_i, width, height, width,
                                    hw_im, val);
        }
        qmulf(val, val, mask);

        *data_col_ptr = val.x;
        data_col_ptr += output_step;
        *data_col_ptr = val.y;
        data_col_ptr += output_step;
        *data_col_ptr = val.z;
        data_col_ptr += output_step;
        *data_col_ptr = val.w;
        data_col_ptr += output_step;
      }
    }
  }
}

template <typename scalar_t>
__global__ void output_add_bias_kernel(scalar_t *output, const scalar_t *bias,
                                       size_t step_batch, size_t step_channel,
                                       size_t n) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    output[index] += bias[(index % step_batch) / step_channel];
  }
}

template <>
__global__ void output_add_bias_kernel(__half *output, const __half *bias,
                                       size_t step_batch, size_t step_channel,
                                       size_t n) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    output[index] =
        __hadd(bias[(index % step_batch) / step_channel], output[index]);
  }
}

template <typename T>
__global__ void output_add_bias_kernel_int8(const int32_t *int32_iw,
                                            float scale_iw, const T *bias,
                                            int8_t *output, float scale_o,
                                            size_t n, int hw_out, int hw4) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int bias_index = index / hw_out;
    const int temp32_index = bias_index * hw4 + index % hw_out;

    output[index] = T2int8<float>(
        (int32_iw[temp32_index] * scale_iw + bias[bias_index]) / scale_o);
  }
}

template <>
__global__ void output_add_bias_kernel_int8(const int32_t *int32_iw,
                                            float scale_iw, const __half *bias,
                                            int8_t *output, float scale_o,
                                            size_t n, int hw_out, int hw4) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int bias_index = index / hw_out;
    const int temp32_index = bias_index * hw4 + index % hw_out;

    output[index] = T2int8<float>(
        (int32_iw[temp32_index] * scale_iw + __half2float(bias[bias_index])) /
        scale_o);
  }
}

__global__ void output_wo_bias_kernel_int8(const int32_t *int32_iw,
                                           float scale_iw, int8_t *output,
                                           float scale_o, size_t n, int hw_out,
                                           int hw4) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int temp32_index = index / hw_out * hw4 + index % hw_out;
    output[index] =
        T2int8<float>((int32_iw[temp32_index] * scale_iw) / scale_o);
  }
}

template <typename scalar_t>
void trt_modulated_deformable_im2col(
    const scalar_t *data_im_, const scalar_t *data_offset_,
    const scalar_t *data_mask_, const int batch_size, const int channels,
    const int height_im, const int width_im, const int height_col,
    const int width_col, const int kernel_h, const int kenerl_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w, const int deformable_group,
    scalar_t *data_col_, hipStream_t stream) {
  // num_axes should be smaller than block size
  const int channel_per_deformable_group = channels / deformable_group;
  const int num_kernels = channels * batch_size * height_col * width_col;

  modulated_deformable_im2col_gpu_kernel<scalar_t>
      <<<GET_BLOCKS(num_kernels), THREADS_PER_BLOCK, 0, stream>>>(
          num_kernels, data_im_, data_offset_, data_mask_, height_im, width_im,
          kernel_h, kenerl_w, pad_h, pad_w, stride_h, stride_w, dilation_h,
          dilation_w, channel_per_deformable_group, batch_size, channels,
          deformable_group, height_col, width_col, data_col_);

  cudaCheckError();
}

void trt_modulated_deformable_im2col_h2(
    const __half2 *data_im_, const __half2 *data_offset_,
    const __half *data_mask_, const int batch_size, const int channels,
    const int height_im, const int width_im, const int height_col,
    const int width_col, const int kernel_h, const int kenerl_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w, const int deformable_group,
    __half2 *data_col_, hipStream_t stream) {
  // num_axes should be smaller than block size
  const int channel_per_deformable_group =
      (channels + 1) / 2 / deformable_group;
  const int num_kernels =
      (channels + 1) / 2 * batch_size * height_col * width_col;

  modulated_deformable_im2col_gpu_kernel_h2<<<GET_BLOCKS(num_kernels),
                                              THREADS_PER_BLOCK, 0, stream>>>(
      num_kernels, data_im_, data_offset_, data_mask_, height_im, width_im,
      kernel_h, kenerl_w, pad_h, pad_w, stride_h, stride_w, dilation_h,
      dilation_w, channel_per_deformable_group, batch_size, channels,
      deformable_group, height_col, width_col, data_col_);

  cudaCheckError();
}

void trt_modulated_deformable_im2col_int8(
    const int8_4 *data_im_, const float &scale_i, const int8_t *data_offset_,
    const float &scale_off, const int8_t *data_mask_, const float &scale_mask,
    const int batch_size, const int channels, const int height_im,
    const int width_im, const int height_col, const int width_col,
    const int kernel_h, const int kenerl_w, const int pad_h, const int pad_w,
    const int stride_h, const int stride_w, const int dilation_h,
    const int dilation_w, const int deformable_group, int8_4 *data_col_,
    hipStream_t stream) {
  // num_axes should be smaller than block size
  const int channel_per_deformable_group =
      (channels + 3) / 4 / deformable_group;
  const int num_kernels =
      (channels + 3) / 4 * batch_size * height_col * width_col;
  const int hw4 = (height_col * width_col + 3) / 4 * 4;

  modulated_deformable_im2col_gpu_kernel_int8<<<GET_BLOCKS(num_kernels),
                                                THREADS_PER_BLOCK, 0, stream>>>(
      num_kernels, data_im_, scale_i, data_offset_, scale_off, data_mask_,
      scale_mask, height_im, width_im, kernel_h, kenerl_w, pad_h, pad_w,
      stride_h, stride_w, dilation_h, dilation_w, channel_per_deformable_group,
      batch_size, channels, deformable_group, height_col, width_col, hw4,
      data_col_);

  cudaCheckError();
}

template <typename scalar_t>
static void output_add_bias(scalar_t *output, const scalar_t *bias,
                            size_t batch, size_t channel, size_t height,
                            size_t width, hipStream_t stream) {
  size_t step_channel = height * width;
  size_t step_batch = step_channel * channel;
  size_t n = step_batch * batch;
  output_add_bias_kernel<scalar_t>
      <<<GET_BLOCKS(n), THREADS_PER_BLOCK, 0, stream>>>(
          output, bias, step_batch, step_channel, n);
}

template <typename scalar_t>
void ModulatedDeformConvForwardCUDAKernel(
    const scalar_t *input, const scalar_t *weight, const scalar_t *bias,
    const scalar_t *offset, const scalar_t *mask, scalar_t *output,
    void *workspace, int batch, int channels, int height, int width,
    int channels_out, int kernel_w, int kernel_h, int stride_w, int stride_h,
    int pad_w, int pad_h, int dilation_w, int dilation_h, int group,
    int deformable_group, int im2col_step, hipblasHandle_t cublas_handle,
    hipStream_t stream) {

  bool with_bias = (bias != nullptr);

  im2col_step = std::min(int(batch), im2col_step);
  ASSERT(batch % im2col_step == 0)

  const int height_out =
      (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
  const int width_out =
      (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;

  auto columns = (scalar_t *)workspace;

  const size_t input_step = channels * height * width;
  const size_t offset_step =
      deformable_group * kernel_h * kernel_w * 2 * height_out * width_out;
  const size_t mask_step =
      deformable_group * kernel_h * kernel_w * height_out * width_out;
  const size_t out_step = channels_out * height_out * width_out;
  const size_t out_group_step = out_step / group;
  const size_t col_g_step =
      channels * kernel_w * kernel_h / group * height_out * width_out;
  const size_t weight_g_step =
      channels_out / group * channels / group * kernel_h * kernel_w;

  const int m = channels_out / group;
  const int n = height_out * width_out;
  const int k = channels / group * kernel_h * kernel_w;
  scalar_t alpha = 1.;
  scalar_t beta = 0.;

  for (int b = 0; b < batch; b++) {
    const scalar_t *input_start = input + b * input_step;
    const scalar_t *offset_start = offset + b * offset_step;
    const scalar_t *mask_start = mask + b * mask_step;
    trt_modulated_deformable_im2col<scalar_t>(
        input_start, offset_start, mask_start, 1, channels, height, width,
        height_out, width_out, kernel_h, kernel_w, pad_h, pad_w, stride_h,
        stride_w, dilation_h, dilation_w, deformable_group, columns, stream);

    for (int g = 0; g < group; g++) {
      const scalar_t *weight_start = weight + g * weight_g_step;
      scalar_t *col_start = columns + g * col_g_step;
      scalar_t *out_buffer_start = output + b * out_step + g * out_group_step;

      cublasGemmWrap<scalar_t>(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                               &alpha, col_start, n, weight_start, k, &beta,
                               out_buffer_start, n);
      cudaCheckError();
    }
  }

  if (with_bias) {
    output_add_bias<scalar_t>(output, bias, batch, channels_out, height_out,
                              width_out, stream);
  }
}

template <>
void ModulatedDeformConvForwardCUDAKernel(
    const __half *input, const __half *weight, const __half *bias,
    const __half *offset, const __half *mask, __half *output, void *workspace,
    int batch, int channels, int height, int width, int channels_out,
    int kernel_w, int kernel_h, int stride_w, int stride_h, int pad_w,
    int pad_h, int dilation_w, int dilation_h, int group, int deformable_group,
    int im2col_step, hipblasHandle_t cublas_handle, hipStream_t stream) {

  bool with_bias = (bias != nullptr);

  im2col_step = std::min(int(batch), im2col_step);
  ASSERT(batch % im2col_step == 0)

  const int height_out =
      (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
  const int width_out =
      (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;

  auto columns = (__half *)workspace;

  const size_t input_step = channels * height * width;
  const size_t offset_step =
      deformable_group * kernel_h * kernel_w * 2 * height_out * width_out;
  const size_t mask_step =
      deformable_group * kernel_h * kernel_w * height_out * width_out;
  const size_t out_step = channels_out * height_out * width_out;
  const size_t out_group_step = out_step / group;
  const size_t col_g_step =
      channels * kernel_w * kernel_h / group * height_out * width_out;
  const size_t weight_g_step =
      channels_out / group * channels / group * kernel_h * kernel_w;

  const int m = channels_out / group;
  const int n = height_out * width_out;
  const int k = channels / group * kernel_h * kernel_w;
  __half alpha = __float2half(1.f);
  __half beta = __float2half(0.f);

  for (int b = 0; b < batch; b++) {
    const __half *input_start = input + b * input_step;
    const __half *offset_start = offset + b * offset_step;
    const __half *mask_start = mask + b * mask_step;
    trt_modulated_deformable_im2col<__half>(
        input_start, offset_start, mask_start, 1, channels, height, width,
        height_out, width_out, kernel_h, kernel_w, pad_h, pad_w, stride_h,
        stride_w, dilation_h, dilation_w, deformable_group, columns, stream);

    for (int g = 0; g < group; g++) {
      const __half *weight_start = weight + g * weight_g_step;
      __half *col_start = columns + g * col_g_step;
      __half *out_buffer_start = output + b * out_step + g * out_group_step;

      cublasGemmWrap<__half>(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                             &alpha, col_start, n, weight_start, k, &beta,
                             out_buffer_start, n);
      cudaCheckError();
    }
  }

  if (with_bias) {
    output_add_bias<__half>(output, bias, batch, channels_out, height_out,
                            width_out, stream);
  }
}

template <>
void ModulatedDeformConvForwardCUDAKernel(
    const __half2 *input, const __half2 *weight, const __half2 *bias,
    const __half2 *offset, const __half2 *mask, __half2 *output,
    void *workspace, int batch, int channels, int height, int width,
    int channels_out, int kernel_w, int kernel_h, int stride_w, int stride_h,
    int pad_w, int pad_h, int dilation_w, int dilation_h, int group,
    int deformable_group, int im2col_step, hipblasHandle_t cublas_handle,
    hipStream_t stream) {

  bool with_bias = (bias != nullptr);

  im2col_step = std::min(int(batch), im2col_step);
  ASSERT(batch % im2col_step == 0)

  const int height_out =
      (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
  const int width_out =
      (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;

  auto columns = (__half2 *)workspace;

  const size_t input_step = (channels + 1) / 2 * height * width;
  const size_t offset_step =
      deformable_group * kernel_h * kernel_w * height_out * width_out;
  const size_t mask_step =
      deformable_group * kernel_h * kernel_w * height_out * width_out;
  const size_t out_step = channels_out * height_out * width_out;
  const size_t out_group_step = out_step / group;
  const size_t col_g_step =
      (channels + 1) / 2 * kernel_w * kernel_h / group * height_out * width_out;
  const size_t weight_g_step =
      channels_out / group * ((channels + 1) / 2) / group * kernel_h * kernel_w;

  const int m = channels_out / group;
  const int n = height_out * width_out;
  const int k = (channels + 1) / 2 / group * kernel_h * kernel_w * 2;
  __half alpha = __float2half(1.f);
  __half beta = __float2half(0.f);

  for (int b = 0; b < batch; b++) {
    const __half2 *input_start = input + b * input_step;
    const __half2 *offset_start = offset + b * offset_step;
    const __half *mask_start = (__half *)mask + b * mask_step;
    trt_modulated_deformable_im2col_h2(
        input_start, offset_start, mask_start, 1, channels, height, width,
        height_out, width_out, kernel_h, kernel_w, pad_h, pad_w, stride_h,
        stride_w, dilation_h, dilation_w, deformable_group, columns, stream);

    for (int g = 0; g < group; g++) {
      const __half2 *weight_start = weight + g * weight_g_step;
      __half2 *col_start = columns + g * col_g_step;
      __half *out_buffer_start =
          (__half *)output + b * out_step + g * out_group_step;

      cublasGemmWrap<__half>(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                             &alpha, (__half *)col_start, n,
                             (__half *)weight_start, k, &beta, out_buffer_start,
                             n);
      cudaCheckError();
    }
  }

  if (with_bias) {
    output_add_bias<__half>((__half *)output, (__half *)bias, batch,
                            channels_out, height_out, width_out, stream);
  }
}

template <typename T>
void ModulatedDeformConvForwardCUDAKernel_int8(
    const int8_4 *input, const float &scale_i, const int8_4 *weight,
    const float &scale_w, const T *bias, const int8_t *offset,
    const float &scale_off, const int8_t *mask, const float &scale_mask,
    int8_t *output, const float &scale_o, void *workspace, int batch,
    int channels, int height, int width, int channels_out, int kernel_w,
    int kernel_h, int stride_w, int stride_h, int pad_w, int pad_h,
    int dilation_w, int dilation_h, int group, int deformable_group,
    int im2col_step, hipblasHandle_t cublas_handle, hipStream_t stream) {

  bool with_bias = (bias != nullptr);

  im2col_step = std::min(int(batch), im2col_step);
  ASSERT(batch % im2col_step == 0)

  const int height_out =
      (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
  const int width_out =
      (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
  const int hw4 = (height_out * width_out + 3) / 4 * 4;
  const int hw_out = height_out * width_out;

  auto columns = (int8_4 *)workspace;
  auto int32_temp =
      (int32_t *)(columns + (channels + 3) / 4 * kernel_w * kernel_h * hw4);

  const size_t input_step = (channels + 3) / 4 * height * width;
  const size_t offset_step =
      deformable_group * kernel_h * kernel_w * height_out * width_out * 2;
  const size_t mask_step =
      deformable_group * kernel_h * kernel_w * height_out * width_out;
  const size_t out_step = channels_out * height_out * width_out;
  const size_t out_group_step = out_step / group;
  const size_t col_g_step =
      (channels + 3) / 4 * kernel_w * kernel_h / group * hw4;
  const size_t weight_g_step =
      channels_out / group * ((channels + 3) / 4) / group * kernel_h * kernel_w;

  const int m = channels_out / group;
  const int n = hw4;
  const int k = (channels + 3) / 4 / group * kernel_h * kernel_w * 4;
  int32_t alpha = 1, beta = 0;
  const float scale_iw = scale_i * scale_w;
  const int output_kernel_count = channels_out / group * height_out * width_out;

  for (int b = 0; b < batch; b++) {
    const int8_4 *input_start = input + b * input_step;
    const int8_t *offset_start = offset + b * offset_step;
    const int8_t *mask_start = mask + b * mask_step;
    trt_modulated_deformable_im2col_int8(
        input_start, scale_i, offset_start, scale_off, mask_start, scale_mask,
        1, channels, height, width, height_out, width_out, kernel_h, kernel_w,
        pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
        deformable_group, columns, stream);

    for (int g = 0; g < group; g++) {
      const int8_4 *weight_start = weight + g * weight_g_step;
      int8_4 *col_start = columns + g * col_g_step;
      int8_t *out_buffer_start = output + b * out_step + g * out_group_step;

      cublasGemmWrap_int8(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                          &alpha, (int8_t *)col_start, n,
                          (int8_t *)weight_start, k, &beta, int32_temp, n);

      if (with_bias) {
        const T *bias_start = bias + g * m;
        output_add_bias_kernel_int8<T>
            <<<GET_BLOCKS(output_kernel_count), THREADS_PER_BLOCK, 0, stream>>>(
                int32_temp, scale_iw, bias_start, out_buffer_start, scale_o,
                output_kernel_count, hw_out, hw4);
      } else {
        output_wo_bias_kernel_int8<<<GET_BLOCKS(output_kernel_count),
                                     THREADS_PER_BLOCK, 0, stream>>>(
            int32_temp, scale_iw, out_buffer_start, scale_o,
            output_kernel_count, hw_out, hw4);
      }

      cudaCheckError();
    }
  }
}

template void ModulatedDeformConvForwardCUDAKernel<float>(
    const float *input, const float *weight, const float *bias,
    const float *offset, const float *mask, float *output, void *workspace,
    int batch, int channels, int height, int width, int channels_out,
    int kernel_w, int kernel_h, int stride_w, int stride_h, int pad_w,
    int pad_h, int dilation_w, int dilation_h, int group, int deformable_group,
    int im2col_step, hipblasHandle_t cublas_handle, hipStream_t stream);

template void ModulatedDeformConvForwardCUDAKernel<__half>(
    const __half *input, const __half *weight, const __half *bias,
    const __half *offset, const __half *mask, __half *output, void *workspace,
    int batch, int channels, int height, int width, int channels_out,
    int kernel_w, int kernel_h, int stride_w, int stride_h, int pad_w,
    int pad_h, int dilation_w, int dilation_h, int group, int deformable_group,
    int im2col_step, hipblasHandle_t cublas_handle, hipStream_t stream);

template void ModulatedDeformConvForwardCUDAKernel<__half2>(
    const __half2 *input, const __half2 *weight, const __half2 *bias,
    const __half2 *offset, const __half2 *mask, __half2 *output,
    void *workspace, int batch, int channels, int height, int width,
    int channels_out, int kernel_w, int kernel_h, int stride_w, int stride_h,
    int pad_w, int pad_h, int dilation_w, int dilation_h, int group,
    int deformable_group, int im2col_step, hipblasHandle_t cublas_handle,
    hipStream_t stream);

template void ModulatedDeformConvForwardCUDAKernel_int8<float>(
    const int8_4 *input, const float &scale_i, const int8_4 *weight,
    const float &scale_w, const float *bias, const int8_t *offset,
    const float &scale_off, const int8_t *mask, const float &scale_mask,
    int8_t *output, const float &scale_o, void *workspace, int batch,
    int channels, int height, int width, int channels_out, int kernel_w,
    int kernel_h, int stride_w, int stride_h, int pad_w, int pad_h,
    int dilation_w, int dilation_h, int group, int deformable_group,
    int im2col_step, hipblasHandle_t cublas_handle, hipStream_t stream);

template void ModulatedDeformConvForwardCUDAKernel_int8<__half>(
    const int8_4 *input, const float &scale_i, const int8_4 *weight,
    const float &scale_w, const __half *bias, const int8_t *offset,
    const float &scale_off, const int8_t *mask, const float &scale_mask,
    int8_t *output, const float &scale_o, void *workspace, int batch,
    int channels, int height, int width, int channels_out, int kernel_w,
    int kernel_h, int stride_w, int stride_h, int pad_w, int pad_h,
    int dilation_w, int dilation_h, int group, int deformable_group,
    int im2col_step, hipblasHandle_t cublas_handle, hipStream_t stream);
