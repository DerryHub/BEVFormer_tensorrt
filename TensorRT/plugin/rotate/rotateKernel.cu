#include "hip/hip_runtime.h"
//
// Created by Derry Lin on 2022/11/21.
//

#include <cstdio>
#include <hip/hip_fp16.h>

#include <algorithm>
#include <cmath>
#include <vector>

#include "cuda_helper.h"
#include "helper.h"
#include "rotateKernel.h"
#include <cstdio>
#include <unistd.h>

template <typename T> __forceinline__ __device__ T sign_05(T x) {
  if (x > 0) {
    return 0.5f;
  }
  return -0.5f;
}

template <typename T> __forceinline__ __device__ int8_t T2int8(T a) {
  a = a > 127 ? 127 : a;
  a = a < -128 ? -128 : a;
  return int8_t(a + sign_05<T>(a));
}

template <> __forceinline__ __device__ int8_t T2int8(__half a) {
  a = __hgt(a, __int2half_rn(127)) ? __int2half_rn(127) : a;
  a = __hlt(a, __int2half_rn(-128)) ? __int2half_rn(-128) : a;
  return int8_t(__half2int_rn(a));
}

__forceinline__ __device__ int8_t half2int8(const __half &hval,
                                            const float &scale) {
  __half ret = __hdiv(hval, __float2half(scale));
  return T2int8<__half>(ret);
}

__forceinline__ __device__ void dp4a(const int32_t *a, const int32_t *b,
                                     int32_t &c) {
#if __CUDA_ARCH__ >= 610
  asm("dp4a.s32.s32 %0, %1, %2, %3;" : "+r"(c) : "r"(*a), "r"(*b), "r"(c));
#else
  auto ap = (int8_4 *)a, bp = (int8_4 *)b;

  c += ap->x * bp->x;
  c += ap->y * bp->y;
  c += ap->z * bp->z;
  c += ap->w * bp->w;
#endif
}

__forceinline__ __device__ void qmulf(const int8_4 &a, int8_4 &c,
                                      const float &b) {
  c.x = T2int8<float>(a.x * b);
  c.y = T2int8<float>(a.y * b);
  c.z = T2int8<float>(a.z * b);
  c.w = T2int8<float>(a.w * b);
}

static __forceinline__ __device__ bool within_bounds_2d(int h, int w, int H,
                                                        int W) {
  return h >= 0 && h < H && w >= 0 && w < W;
}

template <typename scalar_t>
__forceinline__ __device__ scalar_t safe_downgrade_to_int_range(scalar_t x) {
  if (x > INT_MAX - 1 || x < INT_MIN || !::isfinite(static_cast<double>(x)))
    return static_cast<scalar_t>(-100.0);
  return x;
}

template <>
__forceinline__ __device__ __half safe_downgrade_to_int_range(__half x) {
  if (__hisinf(x))
    return static_cast<__half>(-100.0);
  return x;
}

template <>
__forceinline__ __device__ __half2 safe_downgrade_to_int_range(__half2 x) {
  __half low, high;

  low = __low2half(x);
  if (__hisinf(low)) {
    low = static_cast<__half>(-100.0);
  }

  high = __high2half(x);
  if (__hisinf(high)) {
    high = static_cast<__half>(-100.0);
  }

  return __halves2half2(low, high);
}

template <typename scalar_t>
__forceinline__ __device__ scalar_t
grid_sampler_compute_source_index(scalar_t coord, int size) {
  coord = ((coord + 1.f) * size - 1) / 2;
  coord = safe_downgrade_to_int_range(coord);
  return coord;
}

template <>
__forceinline__ __device__ __half
grid_sampler_compute_source_index(__half coord, int size) {
  coord = __hfma(__hfma(coord, __float2half(0.5), __float2half(0.5)),
                 __float2half(static_cast<float>(size)), __float2half(-0.5));
  coord = safe_downgrade_to_int_range(coord);
  return coord;
}

__forceinline__ __device__ __half2
grid_sampler_compute_source_index_h2(__half2 coord, __half2 wh) {
  coord =
      __hfma2(__hfma2(coord, __float2half2_rn(0.5f), __float2half2_rn(0.5f)),
              wh, __float2half2_rn(-0.5f));
  coord = safe_downgrade_to_int_range(coord);
  return coord;
}

template <typename scalar_t>
__global__ void rotateKernel(const int nthreads, scalar_t *output,
                             const scalar_t *input, const scalar_t *angle,
                             const scalar_t *center, int channel, int height,
                             int width, RotateInterpolation interp) {
  int inp_sC = width * height;
  int inp_sH = width;
  int inp_sW = 1;

  const scalar_t ang = -(*angle) * M_PI / 180.f;
  const scalar_t cx = center[0] - 0.5f * width, cy = center[1] - 0.5f * height;
  const scalar_t matrix[6] = {std::cos(ang),
                              std::sin(ang),
                              -cx * std::cos(ang) - cy * std::sin(ang) + cx,
                              -std::sin(ang),
                              std::cos(ang),
                              cx * std::sin(ang) - cy * std::cos(ang) + cy};

  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int w = index % width;
    const int h = (index / width) % height;
    const scalar_t x = -width * 0.5f + 0.5f + w, y = -height * 0.5f + 0.5f + h;

    const scalar_t grid_x =
        (matrix[0] * x + matrix[1] * y + matrix[2]) / (0.5f * width);
    const scalar_t grid_y =
        (matrix[3] * x + matrix[4] * y + matrix[5]) / (0.5f * height);

    scalar_t ix = grid_sampler_compute_source_index(grid_x, width);
    scalar_t iy = grid_sampler_compute_source_index(grid_y, height);

    if (interp == RotateInterpolation::Bilinear) {
      // get NE, NW, SE, SW pixel values from (x, y)
      int ix_nw = static_cast<int>(::floor(ix));
      int iy_nw = static_cast<int>(::floor(iy));
      int ix_ne = ix_nw + 1;
      int iy_ne = iy_nw;
      int ix_sw = ix_nw;
      int iy_sw = iy_nw + 1;
      int ix_se = ix_nw + 1;
      int iy_se = iy_nw + 1;

      // get surfaces to each neighbor:
      scalar_t nw = (ix_se - ix) * (iy_se - iy);
      scalar_t ne = (ix - ix_sw) * (iy_sw - iy);
      scalar_t sw = (ix_ne - ix) * (iy - iy_ne);
      scalar_t se = (ix - ix_nw) * (iy - iy_nw);

      // calculate bilinear weighted pixel value and set output pixel
      auto inp_ptr = input;
      auto out_ptr = output + h * inp_sH + w * inp_sW;
      for (int c = 0; c < channel; ++c, inp_ptr += inp_sC, out_ptr += inp_sC) {
        *out_ptr = static_cast<scalar_t>(0);
        if (within_bounds_2d(iy_nw, ix_nw, height, width)) {
          *out_ptr += inp_ptr[iy_nw * inp_sH + ix_nw * inp_sW] * nw;
        }
        if (within_bounds_2d(iy_ne, ix_ne, height, width)) {
          *out_ptr += inp_ptr[iy_ne * inp_sH + ix_ne * inp_sW] * ne;
        }
        if (within_bounds_2d(iy_sw, ix_sw, height, width)) {
          *out_ptr += inp_ptr[iy_sw * inp_sH + ix_sw * inp_sW] * sw;
        }
        if (within_bounds_2d(iy_se, ix_se, height, width)) {
          *out_ptr += inp_ptr[iy_se * inp_sH + ix_se * inp_sW] * se;
        }
      }
    } else if (interp == RotateInterpolation::Nearest) {
      int ix_nearest = static_cast<int>(::round(ix));
      int iy_nearest = static_cast<int>(::round(iy));

      // assign nearest neighbor pixel value to output pixel
      auto inp_ptr = input;
      auto out_ptr = output + h * inp_sH + w * inp_sW;
      for (int c = 0; c < channel; ++c, inp_ptr += inp_sC, out_ptr += inp_sC) {
        if (within_bounds_2d(iy_nearest, ix_nearest, height, width)) {
          *out_ptr = inp_ptr[iy_nearest * inp_sH + ix_nearest * inp_sW];
        } else {
          *out_ptr = static_cast<scalar_t>(0);
        }
      }
    }
  }
}

template <>
__global__ void rotateKernel(const int nthreads, __half *output,
                             const __half *input, const __half *angle,
                             const __half *center, int channel, int height,
                             int width, RotateInterpolation interp) {
  int inp_sC = width * height;
  int inp_sH = width;
  int inp_sW = 1;

  const __half ang = __hmul(*angle, __float2half(-M_PI / 180.f));
  const __half cx = __hsub(center[0], __float2half(0.5f * width)),
               cy = __hsub(center[1], __float2half(0.5f * height));
  const __half matrix[6] = {
      hcos(ang),  hsin(ang), -cx * hcos(ang) - cy * hsin(ang) + cx,
      -hsin(ang), hcos(ang), cx * hsin(ang) - cy * hcos(ang) + cy};

  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int w = index % width;
    const int h = (index / width) % height;

    const __half x = __hadd(__hfma(__float2half(-0.5f),
                                   __float2half(static_cast<float>(width)),
                                   __float2half(0.5f)),
                            __float2half(static_cast<float>(w)));
    const __half y = __hadd(__hfma(__float2half(-0.5f),
                                   __float2half(static_cast<float>(height)),
                                   __float2half(0.5f)),
                            __float2half(static_cast<float>(h)));

    const __half grid_x = __hdiv(
        __hadd(__hmul(matrix[0], x), __hfma(matrix[1], y, matrix[2])),
        __hmul(__float2half(0.5f), __float2half(static_cast<float>(width))));
    const __half grid_y = __hdiv(
        __hadd(__hmul(matrix[3], x), __hfma(matrix[4], y, matrix[5])),
        __hmul(__float2half(0.5f), __float2half(static_cast<float>(height))));

    __half ix = grid_sampler_compute_source_index(grid_x, width);
    __half iy = grid_sampler_compute_source_index(grid_y, height);

    if (interp == RotateInterpolation::Bilinear) {
      // get NE, NW, SE, SW pixel values from (x, y)
      int ix_nw = static_cast<int>(hfloor(ix));
      int iy_nw = static_cast<int>(hfloor(iy));
      int ix_ne = ix_nw + 1;
      int iy_ne = iy_nw;
      int ix_sw = ix_nw;
      int iy_sw = iy_nw + 1;
      int ix_se = ix_nw + 1;
      int iy_se = iy_nw + 1;

      // get surfaces to each neighbor:
      __half nw = __hmul(__hsub(static_cast<__half>(ix_se), ix),
                         __hsub(static_cast<__half>(iy_se), iy));
      __half ne = __hmul(__hsub(ix, static_cast<__half>(ix_sw)),
                         __hsub(static_cast<__half>(iy_sw), iy));
      __half sw = __hmul(__hsub(static_cast<__half>(ix_ne), ix),
                         __hsub(iy, static_cast<__half>(iy_ne)));
      __half se = __hmul(__hsub(ix, static_cast<__half>(ix_nw)),
                         __hsub(iy, static_cast<__half>(iy_nw)));

      // calculate bilinear weighted pixel value and set output pixel
      auto inp_ptr = input;
      auto out_ptr = output + h * inp_sH + w * inp_sW;
      for (int c = 0; c < channel; ++c, inp_ptr += inp_sC, out_ptr += inp_sC) {
        *out_ptr = static_cast<__half>(0);
        if (within_bounds_2d(iy_nw, ix_nw, height, width)) {
          *out_ptr =
              __hfma(inp_ptr[iy_nw * inp_sH + ix_nw * inp_sW], nw, *out_ptr);
        }
        if (within_bounds_2d(iy_ne, ix_ne, height, width)) {
          *out_ptr =
              __hfma(inp_ptr[iy_ne * inp_sH + ix_ne * inp_sW], ne, *out_ptr);
        }
        if (within_bounds_2d(iy_sw, ix_sw, height, width)) {
          *out_ptr =
              __hfma(inp_ptr[iy_sw * inp_sH + ix_sw * inp_sW], sw, *out_ptr);
        }
        if (within_bounds_2d(iy_se, ix_se, height, width)) {
          *out_ptr =
              __hfma(inp_ptr[iy_se * inp_sH + ix_se * inp_sW], se, *out_ptr);
        }
      }
    } else if (interp == RotateInterpolation::Nearest) {
      int ix_nearest = static_cast<int>(hrint(ix));
      int iy_nearest = static_cast<int>(hrint(iy));

      // assign nearest neighbor pixel value to output pixel
      auto inp_ptr = input;
      auto out_ptr = output + h * inp_sH + w * inp_sW;
      for (int c = 0; c < channel; ++c, inp_ptr += inp_sC, out_ptr += inp_sC) {
        if (within_bounds_2d(iy_nearest, ix_nearest, height, width)) {
          *out_ptr = inp_ptr[iy_nearest * inp_sH + ix_nearest * inp_sW];
        } else {
          *out_ptr = static_cast<__half>(0);
        }
      }
    }
  }
}

__global__ void rotateKernel_h2(const int nthreads, __half2 *output,
                                __half2 *input, __half *angle, __half *center,
                                int channel, int height, int width,
                                RotateInterpolation interp) {
  int inp_sC = width * height;
  int inp_sH = width;
  int inp_sW = 1;

  const __half ang = __hmul(*angle, __float2half(-M_PI / 180.f));
  const __half cx = __hsub(center[0], __float2half(0.5f * width)),
               cy = __hsub(center[1], __float2half(0.5f * height));
  const __half matrix[6] = {
      hcos(ang),  hsin(ang), -cx * hcos(ang) - cy * hsin(ang) + cx,
      -hsin(ang), hcos(ang), cx * hsin(ang) - cy * hcos(ang) + cy};

  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    channel = (channel + 1) / 2;
    const int w = index % width;
    const int h = (index / width) % height;

    const __half2 xy = __hadd2(
        __hfma2(__float2half2_rn(-0.5f),
                __floats2half2_rn(static_cast<float>(width),
                                  static_cast<float>(height)),
                __float2half2_rn(0.5f)),
        __floats2half2_rn(static_cast<float>(w), static_cast<float>(h)));
    const __half2 grid_xy =
        __h2div(__hadd2(__hmul2(__halves2half2(matrix[0], matrix[3]),
                                __half2half2(__low2half(xy))),
                        __hfma2(__halves2half2(matrix[1], matrix[4]),
                                __half2half2(__high2half(xy)),
                                __halves2half2(matrix[2], matrix[5]))),
                __hmul2(__float2half2_rn(0.5f),
                        __floats2half2_rn(static_cast<float>(width),
                                          static_cast<float>(height))));

    __half2 ixy = grid_sampler_compute_source_index_h2(
        grid_xy, __floats2half2_rn(static_cast<float>(width),
                                   static_cast<float>(height)));
    __half ix = __low2half(ixy);
    __half iy = __high2half(ixy);

    if (interp == RotateInterpolation::Bilinear) {
      // get NE, NW, SE, SW pixel values from (x, y)
      int ix_nw = static_cast<int>(hfloor(ix));
      int iy_nw = static_cast<int>(hfloor(iy));
      int ix_ne = ix_nw + 1;
      int iy_ne = iy_nw;
      int ix_sw = ix_nw;
      int iy_sw = iy_nw + 1;
      int ix_se = ix_nw + 1;
      int iy_se = iy_nw + 1;

      // get surfaces to each neighbor:
      __half2 nw = __half2half2(__hmul(__hsub(static_cast<__half>(ix_se), ix),
                                       __hsub(static_cast<__half>(iy_se), iy)));
      __half2 ne = __half2half2(__hmul(__hsub(ix, static_cast<__half>(ix_sw)),
                                       __hsub(static_cast<__half>(iy_sw), iy)));
      __half2 sw = __half2half2(__hmul(__hsub(static_cast<__half>(ix_ne), ix),
                                       __hsub(iy, static_cast<__half>(iy_ne))));
      __half2 se = __half2half2(__hmul(__hsub(ix, static_cast<__half>(ix_nw)),
                                       __hsub(iy, static_cast<__half>(iy_nw))));

      // calculate bilinear weighted pixel value and set output pixel
      auto inp_ptr = input;
      auto out_ptr = output + h * inp_sH + w * inp_sW;
      for (int c = 0; c < channel; ++c, inp_ptr += inp_sC, out_ptr += inp_sC) {
        *out_ptr = __float2half2_rn(0.f);
        if (within_bounds_2d(iy_nw, ix_nw, height, width)) {
          *out_ptr =
              __hfma2(inp_ptr[iy_nw * inp_sH + ix_nw * inp_sW], nw, *out_ptr);
        }
        if (within_bounds_2d(iy_ne, ix_ne, height, width)) {
          *out_ptr =
              __hfma2(inp_ptr[iy_ne * inp_sH + ix_ne * inp_sW], ne, *out_ptr);
        }
        if (within_bounds_2d(iy_sw, ix_sw, height, width)) {
          *out_ptr =
              __hfma2(inp_ptr[iy_sw * inp_sH + ix_sw * inp_sW], sw, *out_ptr);
        }
        if (within_bounds_2d(iy_se, ix_se, height, width)) {
          *out_ptr =
              __hfma2(inp_ptr[iy_se * inp_sH + ix_se * inp_sW], se, *out_ptr);
        }
      }
    } else if (interp == RotateInterpolation::Nearest) {
      int ix_nearest = static_cast<int>(hrint(ix));
      int iy_nearest = static_cast<int>(hrint(iy));

      // assign nearest neighbor pixel value to output pixel
      auto inp_ptr = input;
      auto out_ptr = output + h * inp_sH + w * inp_sW;
      for (int c = 0; c < channel; ++c, inp_ptr += inp_sC, out_ptr += inp_sC) {
        if (within_bounds_2d(iy_nearest, ix_nearest, height, width)) {
          *out_ptr = inp_ptr[iy_nearest * inp_sH + ix_nearest * inp_sW];
        } else {
          *out_ptr = __float2half2_rn(0.f);
        }
      }
    }
  }
}

template <typename T>
__global__ void rotateKernel_int8(const int nthreads, int8_4 *output,
                                  float scale_o, const int8_4 *input,
                                  float scale_i, const T *angle,
                                  const T *center, int channel, int height,
                                  int width, RotateInterpolation interp) {
  int inp_sC = width * height;
  int inp_sH = width;
  int inp_sW = 1;

  const __half ang = __hmul(__float2half(*angle), __float2half(-M_PI / 180.f));
  const __half cx = __hsub(__float2half(center[0]), __float2half(0.5f * width)),
               cy =
                   __hsub(__float2half(center[1]), __float2half(0.5f * height));
  const __half matrix[6] = {
      hcos(ang),  hsin(ang), -cx * hcos(ang) - cy * hsin(ang) + cx,
      -hsin(ang), hcos(ang), cx * hsin(ang) - cy * hcos(ang) + cy};

  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    channel = (channel + 3) / 4;
    const int w = index % width;
    const int h = (index / width) % height;

    const __half2 xy = __hadd2(
        __hfma2(__float2half2_rn(-0.5f),
                __floats2half2_rn(static_cast<float>(width),
                                  static_cast<float>(height)),
                __float2half2_rn(0.5f)),
        __floats2half2_rn(static_cast<float>(w), static_cast<float>(h)));
    const __half2 grid_xy =
        __h2div(__hadd2(__hmul2(__halves2half2(matrix[0], matrix[3]),
                                __half2half2(__low2half(xy))),
                        __hfma2(__halves2half2(matrix[1], matrix[4]),
                                __half2half2(__high2half(xy)),
                                __halves2half2(matrix[2], matrix[5]))),
                __hmul2(__float2half2_rn(0.5f),
                        __floats2half2_rn(static_cast<float>(width),
                                          static_cast<float>(height))));

    __half2 ixy = grid_sampler_compute_source_index_h2(
        grid_xy, __floats2half2_rn(static_cast<float>(width),
                                   static_cast<float>(height)));
    __half ix = __low2half(ixy);
    __half iy = __high2half(ixy);

    if (interp == RotateInterpolation::Bilinear) {
      // get NE, NW, SE, SW pixel values from (x, y)
      int ix_nw = static_cast<int>(hfloor(ix));
      int iy_nw = static_cast<int>(hfloor(iy));
      int ix_ne = ix_nw + 1;
      int iy_ne = iy_nw;
      int ix_sw = ix_nw;
      int iy_sw = iy_nw + 1;
      int ix_se = ix_nw + 1;
      int iy_se = iy_nw + 1;

      // get surfaces to each neighbor:
      float scale_area = 1 / 127.f;
      float scale_out = scale_area * scale_i / scale_o;
      int8_4 weight;
      weight.x = half2int8(__hmul(__hsub(static_cast<__half>(ix_se), ix),
                                  __hsub(static_cast<__half>(iy_se), iy)),
                           scale_area);
      weight.y = half2int8(__hmul(__hsub(ix, static_cast<__half>(ix_sw)),
                                  __hsub(static_cast<__half>(iy_sw), iy)),
                           scale_area);
      weight.z = half2int8(__hmul(__hsub(static_cast<__half>(ix_ne), ix),
                                  __hsub(iy, static_cast<__half>(iy_ne))),
                           scale_area);
      weight.w = half2int8(__hmul(__hsub(ix, static_cast<__half>(ix_nw)),
                                  __hsub(iy, static_cast<__half>(iy_nw))),
                           scale_area);
      int8_4 inps[4];
      int32_t output_temp;

      // calculate bilinear weighted pixel value and set output pixel
      auto inp_ptr = input;
      auto out_ptr = output + h * inp_sH + w * inp_sW;
      for (int c = 0; c < channel; ++c, inp_ptr += inp_sC, out_ptr += inp_sC) {
        if (within_bounds_2d(iy_nw, ix_nw, height, width)) {
          const int8_4 &inp = inp_ptr[iy_nw * inp_sH + ix_nw * inp_sW];
          inps[0].x = inp.x;
          inps[1].x = inp.y;
          inps[2].x = inp.z;
          inps[3].x = inp.w;
        }
        if (within_bounds_2d(iy_ne, ix_ne, height, width)) {
          const int8_4 &inp = inp_ptr[iy_ne * inp_sH + ix_ne * inp_sW];
          inps[0].y = inp.x;
          inps[1].y = inp.y;
          inps[2].y = inp.z;
          inps[3].y = inp.w;
        }
        if (within_bounds_2d(iy_sw, ix_sw, height, width)) {
          const int8_4 &inp = inp_ptr[iy_sw * inp_sH + ix_sw * inp_sW];
          inps[0].z = inp.x;
          inps[1].z = inp.y;
          inps[2].z = inp.z;
          inps[3].z = inp.w;
        }
        if (within_bounds_2d(iy_se, ix_se, height, width)) {
          const int8_4 &inp = inp_ptr[iy_se * inp_sH + ix_se * inp_sW];
          inps[0].w = inp.x;
          inps[1].w = inp.y;
          inps[2].w = inp.z;
          inps[3].w = inp.w;
        }
        output_temp = 0;
        dp4a((const int32_t *)inps, (const int32_t *)&weight, output_temp);
        out_ptr->x = T2int8<float>(output_temp * scale_out);

        output_temp = 0;
        dp4a((const int32_t *)(inps + 1), (const int32_t *)&weight,
             output_temp);
        out_ptr->y = T2int8<float>(output_temp * scale_out);

        output_temp = 0;
        dp4a((const int32_t *)(inps + 2), (const int32_t *)&weight,
             output_temp);
        out_ptr->z = T2int8<float>(output_temp * scale_out);

        output_temp = 0;
        dp4a((const int32_t *)(inps + 3), (const int32_t *)&weight,
             output_temp);
        out_ptr->w = T2int8<float>(output_temp * scale_out);
      }
    } else if (interp == RotateInterpolation::Nearest) {
      int ix_nearest = static_cast<int>(hrint(ix));
      int iy_nearest = static_cast<int>(hrint(iy));
      float scale_out = scale_i / scale_o;

      // assign nearest neighbor pixel value to output pixel
      auto inp_ptr = input;
      auto out_ptr = output + h * inp_sH + w * inp_sW;
      for (int c = 0; c < channel; ++c, inp_ptr += inp_sC, out_ptr += inp_sC) {
        if (within_bounds_2d(iy_nearest, ix_nearest, height, width)) {
          const int8_4 &inp =
              inp_ptr[iy_nearest * inp_sH + ix_nearest * inp_sW];
          qmulf(inp, *out_ptr, scale_out);
        } else {
          *out_ptr = 0;
        }
      }
    }
  }
}

template <>
__global__ void rotateKernel_int8(const int nthreads, int8_4 *output,
                                  float scale_o, const int8_4 *input,
                                  float scale_i, const __half *angle,
                                  const __half *center, int channel, int height,
                                  int width, RotateInterpolation interp) {
  int inp_sC = width * height;
  int inp_sH = width;
  int inp_sW = 1;

  const __half ang = __hmul(*angle, __float2half(-M_PI / 180.f));
  const __half cx = __hsub(center[0], __float2half(0.5f * width)),
               cy = __hsub(center[1], __float2half(0.5f * height));
  const __half matrix[6] = {
      hcos(ang),  hsin(ang), -cx * hcos(ang) - cy * hsin(ang) + cx,
      -hsin(ang), hcos(ang), cx * hsin(ang) - cy * hcos(ang) + cy};

  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    channel = (channel + 3) / 4;
    const int w = index % width;
    const int h = (index / width) % height;

    const __half2 xy = __hadd2(
        __hfma2(__float2half2_rn(-0.5f),
                __floats2half2_rn(static_cast<float>(width),
                                  static_cast<float>(height)),
                __float2half2_rn(0.5f)),
        __floats2half2_rn(static_cast<float>(w), static_cast<float>(h)));
    const __half2 grid_xy =
        __h2div(__hadd2(__hmul2(__halves2half2(matrix[0], matrix[3]),
                                __half2half2(__low2half(xy))),
                        __hfma2(__halves2half2(matrix[1], matrix[4]),
                                __half2half2(__high2half(xy)),
                                __halves2half2(matrix[2], matrix[5]))),
                __hmul2(__float2half2_rn(0.5f),
                        __floats2half2_rn(static_cast<float>(width),
                                          static_cast<float>(height))));

    __half2 ixy = grid_sampler_compute_source_index_h2(
        grid_xy, __floats2half2_rn(static_cast<float>(width),
                                   static_cast<float>(height)));
    __half ix = __low2half(ixy);
    __half iy = __high2half(ixy);

    if (interp == RotateInterpolation::Bilinear) {
      // get NE, NW, SE, SW pixel values from (x, y)
      int ix_nw = static_cast<int>(hfloor(ix));
      int iy_nw = static_cast<int>(hfloor(iy));
      int ix_ne = ix_nw + 1;
      int iy_ne = iy_nw;
      int ix_sw = ix_nw;
      int iy_sw = iy_nw + 1;
      int ix_se = ix_nw + 1;
      int iy_se = iy_nw + 1;

      // get surfaces to each neighbor:
      float scale_area = 1 / 127.f;
      float scale_out = scale_area * scale_i / scale_o;
      int8_4 weight;
      weight.x = half2int8(__hmul(__hsub(static_cast<__half>(ix_se), ix),
                                  __hsub(static_cast<__half>(iy_se), iy)),
                           scale_area);
      weight.y = half2int8(__hmul(__hsub(ix, static_cast<__half>(ix_sw)),
                                  __hsub(static_cast<__half>(iy_sw), iy)),
                           scale_area);
      weight.z = half2int8(__hmul(__hsub(static_cast<__half>(ix_ne), ix),
                                  __hsub(iy, static_cast<__half>(iy_ne))),
                           scale_area);
      weight.w = half2int8(__hmul(__hsub(ix, static_cast<__half>(ix_nw)),
                                  __hsub(iy, static_cast<__half>(iy_nw))),
                           scale_area);
      int8_4 inps[4];
      int32_t output_temp;

      // calculate bilinear weighted pixel value and set output pixel
      auto inp_ptr = input;
      auto out_ptr = output + h * inp_sH + w * inp_sW;
      for (int c = 0; c < channel; ++c, inp_ptr += inp_sC, out_ptr += inp_sC) {
        if (within_bounds_2d(iy_nw, ix_nw, height, width)) {
          const int8_4 &inp = inp_ptr[iy_nw * inp_sH + ix_nw * inp_sW];
          inps[0].x = inp.x;
          inps[1].x = inp.y;
          inps[2].x = inp.z;
          inps[3].x = inp.w;
        }
        if (within_bounds_2d(iy_ne, ix_ne, height, width)) {
          const int8_4 &inp = inp_ptr[iy_ne * inp_sH + ix_ne * inp_sW];
          inps[0].y = inp.x;
          inps[1].y = inp.y;
          inps[2].y = inp.z;
          inps[3].y = inp.w;
        }
        if (within_bounds_2d(iy_sw, ix_sw, height, width)) {
          const int8_4 &inp = inp_ptr[iy_sw * inp_sH + ix_sw * inp_sW];
          inps[0].z = inp.x;
          inps[1].z = inp.y;
          inps[2].z = inp.z;
          inps[3].z = inp.w;
        }
        if (within_bounds_2d(iy_se, ix_se, height, width)) {
          const int8_4 &inp = inp_ptr[iy_se * inp_sH + ix_se * inp_sW];
          inps[0].w = inp.x;
          inps[1].w = inp.y;
          inps[2].w = inp.z;
          inps[3].w = inp.w;
        }
        output_temp = 0;
        dp4a((const int32_t *)inps, (const int32_t *)&weight, output_temp);
        out_ptr->x = T2int8<float>(output_temp * scale_out);

        output_temp = 0;
        dp4a((const int32_t *)(inps + 1), (const int32_t *)&weight,
             output_temp);
        out_ptr->y = T2int8<float>(output_temp * scale_out);

        output_temp = 0;
        dp4a((const int32_t *)(inps + 2), (const int32_t *)&weight,
             output_temp);
        out_ptr->z = T2int8<float>(output_temp * scale_out);

        output_temp = 0;
        dp4a((const int32_t *)(inps + 3), (const int32_t *)&weight,
             output_temp);
        out_ptr->w = T2int8<float>(output_temp * scale_out);
      }
    } else if (interp == RotateInterpolation::Nearest) {
      int ix_nearest = static_cast<int>(hrint(ix));
      int iy_nearest = static_cast<int>(hrint(iy));
      float scale_out = scale_i / scale_o;

      // assign nearest neighbor pixel value to output pixel
      auto inp_ptr = input;
      auto out_ptr = output + h * inp_sH + w * inp_sW;
      for (int c = 0; c < channel; ++c, inp_ptr += inp_sC, out_ptr += inp_sC) {
        if (within_bounds_2d(iy_nearest, ix_nearest, height, width)) {
          const int8_4 &inp =
              inp_ptr[iy_nearest * inp_sH + ix_nearest * inp_sW];
          qmulf(inp, *out_ptr, scale_out);
        } else {
          *out_ptr = 0;
        }
      }
    }
  }
}

template <typename T>
void rotate(T *output, T *input, T *angle, T *center, int *input_dims,
            RotateInterpolation interp, hipStream_t stream) {
  int channel = input_dims[0];
  int height = input_dims[1];
  int width = input_dims[2];

  int count = height * width;

  rotateKernel<<<GET_BLOCKS(count), THREADS_PER_BLOCK, 0, stream>>>(
      count, output, input, angle, center, channel, height, width, interp);
}

void rotate_h2(__half2 *output, __half2 *input, __half *angle, __half *center,
               int *input_dims, RotateInterpolation interp,
               hipStream_t stream) {
  int channel = input_dims[0];
  int height = input_dims[1];
  int width = input_dims[2];

  int count = height * width;

  rotateKernel_h2<<<GET_BLOCKS(count), THREADS_PER_BLOCK, 0, stream>>>(
      count, output, input, angle, center, channel, height, width, interp);
}

template <typename T>
void rotate_int8(int8_4 *output, float scale_o, const int8_4 *input,
                 float scale_i, const T *angle, const T *center,
                 int *input_dims, RotateInterpolation interp,
                 hipStream_t stream) {
  int channel = input_dims[0];
  int height = input_dims[1];
  int width = input_dims[2];

  int count = height * width;

  rotateKernel_int8<<<GET_BLOCKS(count), THREADS_PER_BLOCK, 0, stream>>>(
      count, output, scale_o, input, scale_i, angle, center, channel, height,
      width, interp);
}

template void rotate(float *output, float *input, float *angle, float *center,
                     int *input_dims, RotateInterpolation interp,
                     hipStream_t stream);

template void rotate(__half *output, __half *input, __half *angle,
                     __half *center, int *input_dims,
                     RotateInterpolation interp, hipStream_t stream);

template void rotate_int8(int8_4 *output, float scale_o, const int8_4 *input,
                          float scale_i, const float *angle,
                          const float *center, int *input_dims,
                          RotateInterpolation interp, hipStream_t stream);

template void rotate_int8(int8_4 *output, float scale_o, const int8_4 *input,
                          float scale_i, const __half *angle,
                          const __half *center, int *input_dims,
                          RotateInterpolation interp, hipStream_t stream);
